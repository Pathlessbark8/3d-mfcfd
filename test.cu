/*  
	MFCFD is a 3D Computational Fluid Dynamics Solver based off q-LSKUM
    Copyright (C) 2022 
    This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <https://www.gnu.org/licenses/>.
*/
// Check read/write permissions
#include <iostream>
#include <fstream>
#include <random>
#include "split_fluxes_mod.h"
#include <iomanip>
#include "octant_fluxes_mod.h"
#include "wall_flux_dGxneg_mod.h"
#include "wall_flux_dGyneg_mod.h"
#include "wall_flux_dGxpos_mod.h"
#include "wall_flux_dGypos_mod.h"
#include "wall_flux_dGzneg_mod.h"
#include "point_preprocessor_mod.h"
#include "compute_conserved_vector_mod.h"
#include "timestep_delt_mod.h"
#include "generate_connectivity_mod.h"
#include "implicit_aliasing_mod.h"
#include "flux_residual_mod.h"
#include "initial_conditions_mod.h"
#include "interior_flux_dGxneg_mod.h"
#include "interior_flux_dGxpos_mod.h"
#include "interior_flux_dGyneg_mod.h"
#include "interior_flux_dGypos_mod.h"
#include <chrono>
#include <hip/hip_runtime.h>
#include "fpi_solver_mod.h"
#include <nccl.h>
#include <mpi.h>
#include <unistd.h>
#include <stdint.h>
#include <stdlib.h>
#include "mpi-ext.h"

// #include "cuPrintf.cu"
using namespace std;
using namespace std::chrono;



static uint64_t getHostHash(const char* string) {
  // Based on DJB2a, result = result * 33 ^ char
  uint64_t result = 5381;
  for (int c = 0; string[c] != '\0'; c++){
    result = ((result << 5) + result) ^ string[c];
  }
  return result;
}


static void getHostName(char* hostname, int maxlen) {
  gethostname(hostname, maxlen);
  for (int i=0; i< maxlen; i++) {
    if (hostname[i] == '.') {
        hostname[i] = '\0';
        return;
    }
  }
}

int main(int argc, char* argv[])
{

    //MAIN CODE BEGINS
    cout<<setprecision(13)<<scientific;

    read_input_point_data();
    initial_conditions();
    generate_split_stencils();
    //
    int myRank, nRanks, localRank = 0;
  
    //initializing MPI
    MPICHECK(MPI_Init(&argc, &argv));
    MPICHECK(MPI_Comm_rank(MPI_COMM_WORLD, &myRank));
    MPICHECK(MPI_Comm_size(MPI_COMM_WORLD, &nRanks));
    //


    //CHECK IF MPI LIBRARY HAD CUDA SUPPORT
    if(myRank==0){
      printf("Compile time check:\n");
      #if defined(MPIX_CUDA_AWARE_SUPPORT) && MPIX_CUDA_AWARE_SUPPORT
          printf("This MPI library has CUDA-aware support.\n", MPIX_CUDA_AWARE_SUPPORT);
      #elif defined(MPIX_CUDA_AWARE_SUPPORT) && !MPIX_CUDA_AWARE_SUPPORT
          printf("This MPI library does not have CUDA-aware support.\n");
      #else
          printf("This MPI library cannot determine if there is CUDA-aware support.\n");
      #endif /* MPIX_CUDA_AWARE_SUPPORT */
      
          printf("Run time check:\n");
      #if defined(MPIX_CUDA_AWARE_SUPPORT)
          if (1 == MPIX_Query_cuda_support()) {
              printf("This MPI library has CUDA-aware support.\n");
          } else {
              printf("This MPI library does not have CUDA-aware support.\n");
          }
      #else /* !defined(MPIX_CUDA_AWARE_SUPPORT) */
          printf("This MPI library cannot determine if there is CUDA-aware support.\n");
      #endif /* MPIX_CUDA_AWARE_SUPPORT */

      cout<<"Total Number of Proccess are "<<nRanks<<endl;
    }
    //
    // HASHING HOSTNAME TO GET LOCALRANKS
    uint64_t hostHashs[nRanks];
    char hostname[1024];
    getHostName(hostname, 1024);
    hostHashs[myRank] = getHostHash(hostname);
    MPICHECK(MPI_Allgather(MPI_IN_PLACE, 0, MPI_DATATYPE_NULL, hostHashs, sizeof(uint64_t), MPI_BYTE, MPI_COMM_WORLD));
    for (int p=0; p<nRanks; p++) {
      if (p == myRank) break;
      if (hostHashs[p] == hostHashs[myRank]) localRank++;
    }
  
    if(myRank==0){
      cout<<"Reading from File\n";
    }
    //READ POINTS FOR EACH DEVICE FROM FILE
    fstream fin;
    fin.open("/home/nsm/3d-mfcfd/inputFiles/"+to_string(max_points)+"/filesFor"+to_string(nRanks)+"Devices/Device"+to_string(myRank)+".dat",ios::in);
    fin>>numDevices;
    fin>>local_points;
    int counter;
    localToGlobalIndex=new int [local_points];
    globalToGhostIndex=new int*[nRanks];
    for(int i=0;i<nRanks;i++)
    {
        globalToGhostIndex[i]=new int[max_points];
    }

    for(int i=0;i<local_points;i++){
        fin>>localToGlobalIndex[i];
        fin>>partVector[localToGlobalIndex[i]];
        globalToLocalIndex[localToGlobalIndex[i]]=i;
        numberOfPointsPerDevice++;
    }
    cout<<"SIZE is "<<numberOfPointsPerDevice * sizeof(splitPoints)<<endl;
    //Share Partition Value across Proccesses
    MPICHECK(MPI_Allreduce(MPI_IN_PLACE, &partVector, max_points, MPI_INT, MPI_SUM, MPI_COMM_WORLD));    

    //ALLOCATING MEMORY FOR POINTS
    splitPoint=new splitPoints[numberOfPointsPerDevice];

    if(myRank==0){
      cout<<"Determining Nature of Points\n";
    }
    //ASSIGNING POINTS FOR EACH DEVICE AND CALCULATING NATURE OF POINTS ON EACH PARTITION
    for(int i=0;i<local_points;i++){
      assign(splitPoint[i],localToGlobalIndex[i],myRank);
      findNatureOfLocalPoints(splitPoint[i]);
      // printf("%d\n",i);
    }
    allocateSizeForNatureOfLocalPoints();
    for(int i=0;i<local_points;i++){
      assignNatureOfLocalPoints(splitPoint[i],i);
    }
    //Initialising the Send Buffer
    sendBuffer=new transferPoints*[nRanks];
    int points_on_gpu_to_send_to;
    int total_points_to_send=0;
    for(int i=0;i<nRanks;i++){
        fin>>points_on_gpu_to_send_to;
        total_points_to_send+=points_on_gpu_to_send_to;
        sendBuffer[i]=new transferPoints[points_on_gpu_to_send_to];
    }
    int currDevice=0;
    int *sendPoints=new int[nRanks];
    for(int i=0;i<nRanks;i++){
        sendPoints[i]=0;
    }

    //Populating the Send Buffer with Data
    for(int i=0;i<total_points_to_send;++i){
        fin>>currDevice;
        fin>>setprecision(13)>>sendBuffer[currDevice][sendPoints[currDevice]].globalIndex>>sendBuffer[currDevice][sendPoints[currDevice]].x>>sendBuffer[currDevice][sendPoints[currDevice]].y>>sendBuffer[currDevice][sendPoints[currDevice]].z;
        int currIndex=globalToLocalIndex[sendBuffer[currDevice][sendPoints[currDevice]].globalIndex];
        splitPoint[currIndex].isGhost=true;
        splitPoint[currIndex].ghostIndex[splitPoint[currIndex].numberOfPartitionsToSendTo]=sendPoints[currDevice];
        splitPoint[currIndex].partitions[splitPoint[currIndex].numberOfPartitionsToSendTo]=currDevice;
        splitPoint[currIndex].numberOfPartitionsToSendTo++;
        globalToGhostIndex[currDevice][sendBuffer[currDevice][sendPoints[currDevice]].globalIndex]=sendPoints[currDevice];
        sendPoints[currDevice]++;   
    }
    fin.close();
    cout<<"Number of Points in Process "<<myRank<<" are: "<<numberOfPointsPerDevice<<endl;
    //Sharing Size across All Processes
    int *receivePoints=new int[nRanks];
    for(int i=0;i<nRanks;i++){
        receivePoints[i]=0;
    }
    for(int i=0;i<nRanks;i++){
      if(i!=myRank){
        MPI_Request request;
        MPICHECK(MPI_Isend(sendPoints+i , 1, MPI_INT, i, 0, MPI_COMM_WORLD,&request));
      }
    }
    for(int i=0;i<nRanks;i++){
      if(i!=myRank){
        // MPICHECK(MPI_Recv(receivePoints+i , 1, MPI_INT, MPI , 0, MPI_COMM_WORLD,MPI_STATUS_IGNORE));
        MPICHECK(MPI_Recv(receivePoints+i , 1, MPI_INT, i, 0, MPI_COMM_WORLD,MPI_STATUS_IGNORE));
      }
    }

    receiveBuffer=new transferPoints*[nRanks];
    for(int i=0;i<nRanks;i++){
      receiveBuffer[i]=new transferPoints[receivePoints[i]];
    }


    //CREATE MPI STRUCTURE TO TRANSFER POINTS DATA TO OTHER PROCESSES
    // const int nitems=5;
    // int          blocklengths[5] = {1,1,1,1,5};
    // MPI_Datatype types[5] = {MPI_INT, MPI_DOUBLE,MPI_DOUBLE,MPI_DOUBLE,MPI_DOUBLE};
    // MPI_Datatype mpi_transferPoints;
    // MPI_Aint     offsets[5];

    // offsets[0] = offsetof(transferPoints, counter);
    // offsets[1] = offsetof(transferPoints, x);
    // offsets[2] = offsetof(transferPoints, y);
    // offsets[3] = offsetof(transferPoints, z);
    // offsets[4] = offsetof(transferPoints, q);


    // MPI_Type_create_struct(nitems, blocklengths, offsets, types, &mpi_transferPoints);
    // MPI_Type_commit(&mpi_transferPoints);
    // //

    int totalPointsToSend=0;
    for(int i=0;i<nRanks;i++){
      totalPointsToSend+=sendPoints[i];
    }

    //Initialising and transfering memory to device Pointers
    splitPoints *splitPoint_d;

    int *globalToLocalIndex_temp;
    int **globalToGhostIndex_send,**globalToGhostIndex_receive;
    int **globalToGhostIndexSendPointer=(int**)malloc(sizeof(int*)*nRanks);
    int** globalToGhostIndexReceivePointer=(int**)malloc(sizeof(int*)*nRanks);
    int *partVector_d;

    transferPoints** sendBuffer_d,** receiveBuffer_d;
    transferPoints** sendPointer=(transferPoints**)malloc(sizeof(transferPoints*)*nRanks);
    transferPoints** receivePointer=(transferPoints**)malloc(sizeof(transferPoints*)*nRanks);
    // hipCtx_t * temp;
    // hipCtxCreate(temp,hipDeviceScheduleAuto,localRank);
    if(myRank==0){
        cout<<"Setting Device"<<endl;
    }
    CUDACHECK(hipSetDevice(localRank));

    //POINTER TO POINTER
    CUDACHECK(hipMalloc(&sendBuffer_d,sizeof(transferPoints*)*nRanks));
    for(int i = 0; i < nRanks; i++)
    {
      transferPoints *darray;
      CUDACHECK(hipMalloc(&darray, sizeof(transferPoints) * sendPoints[i]));
      CUDACHECK(hipMemcpy(darray,sendBuffer[i], sendPoints[i] * sizeof(transferPoints), hipMemcpyHostToDevice));
      CUDACHECK(hipMemcpy(&sendBuffer_d[i], &darray, sizeof(transferPoints*), hipMemcpyHostToDevice));
      memcpy(&sendPointer[i],&darray,sizeof(transferPoints*));
    }

    CUDACHECK(hipMalloc(&receiveBuffer_d,sizeof(transferPoints*)*nRanks));
    for(int i = 0; i < nRanks; i++)
    {
      transferPoints *darray;
      CUDACHECK(hipMalloc(&darray, sizeof(transferPoints) * receivePoints[i]));
      CUDACHECK(hipMemcpy(&receiveBuffer_d[i], &darray, sizeof(transferPoints*), hipMemcpyHostToDevice));
      memcpy(&receivePointer[i],&darray,sizeof(transferPoints*));
    }

    CUDACHECK(hipMalloc(&globalToGhostIndex_send, nRanks * sizeof(int*)));
    for(int i = 0; i < nRanks; i++)
    {
      int *darray;
      CUDACHECK(hipMalloc(&darray, sizeof(int) * max_points));
      CUDACHECK(hipMemcpy(darray,globalToGhostIndex[i], sizeof(int) * max_points, hipMemcpyHostToDevice));
      CUDACHECK(hipMemcpy(&globalToGhostIndex_send[i], &darray, sizeof(int*), hipMemcpyHostToDevice));
      memcpy(&globalToGhostIndexSendPointer[i],&darray,sizeof(int*));
    }

    CUDACHECK(hipMalloc(&globalToGhostIndex_receive, nRanks * sizeof(int*)));
    for(int i = 0; i < nRanks; i++)
    {
        int *darray;
        CUDACHECK(hipMalloc(&darray, sizeof(int) * max_points));
        CUDACHECK(hipMemcpy(&globalToGhostIndex_receive[i], &darray, sizeof(int*), hipMemcpyHostToDevice));
        memcpy(&globalToGhostIndexReceivePointer[i],&darray,sizeof(int*));
    }

    if(myRank==0){
        cout<<"Copying linear arrays"<<endl;
    }
    CUDACHECK(hipMalloc(&splitPoint_d, numberOfPointsPerDevice * sizeof(splitPoints)));
    CUDACHECK(hipMemcpy(splitPoint_d, splitPoint, numberOfPointsPerDevice * sizeof(splitPoints), hipMemcpyHostToDevice));
    CUDACHECK(hipMalloc(&globalToLocalIndex_temp, max_points * sizeof(int)));
    CUDACHECK(hipMemcpy(globalToLocalIndex_temp, globalToLocalIndex, max_points * sizeof(int), hipMemcpyHostToDevice));
    CUDACHECK(hipMalloc(&partVector_d, max_points * sizeof(int)));
    CUDACHECK(hipMemcpy(partVector_d, &partVector, max_points * sizeof(int), hipMemcpyHostToDevice));
    
    // //
   
    
    if (myRank == 0) {
      cout<<"Getting NCCL Unique ID\n";
      NCCLCHECK(ncclGetUniqueId(&id));
    }
    MPICHECK(MPI_Barrier(MPI_COMM_WORLD));

    MPICHECK(MPI_Bcast((void *)&id, sizeof(id), MPI_BYTE, 0, MPI_COMM_WORLD));
    // //
    if(myRank==0){
      cout<<"ID Succesfully Broadcasted\n";
    }

    // Initialising NCCL Communicator and CUDA stream
    ncclComm_t comm;
    NCCLCHECK(ncclCommInitRank(&comm, nRanks, id, myRank));
    hipStream_t stream;

    if(myRank==0){
      cout<<"Beginning Solver\n";
    }

    auto start = high_resolution_clock::now();
    // 
    fpi_solver_multi_nccl(splitPoint_d,localRank,sendBuffer_d,receiveBuffer_d,nRanks,myRank,sendPoints,receivePoints,comm,stream,sendPointer,receivePointer,globalToLocalIndex_temp,globalToGhostIndex_receive,globalToGhostIndexSendPointer,globalToGhostIndexReceivePointer,partVector_d);
    // 
    auto stop = high_resolution_clock::now();
    if(myRank==0){
      cout<<"Copying memory back to Host\n";
    }
    // CUDACHECK(hipMemcpy(splitPoint, splitPoint_d, numberOfPointsPerDevice * sizeof(splitPoints), hipMemcpyDeviceToHost));

    
    // TO COPY BACK THE SEND BUFFER TO HOST (POINTER TO POINTER METHOD)
    // transferPoints *darray;
    // for (int i = 0; i <nRanks; i++){
    //   hipMalloc(&darray, sizeof(transferPoints) * sendPoints[i]);
    //   hipMemcpy(&darray, &sendBuffer_d[i], sizeof(transferPoints*), hipMemcpyDeviceToHost);
    //   hipMemcpy(sendBuffer[i], darray, sizeof(transferPoints) * sendPoints[i], hipMemcpyDeviceToHost);
    //   hipFree(darray);
    // }

    // for (int i = 0; i <nRanks; i++){
    //   hipMalloc(&darray, sizeof(transferPoints) * receivePoints[i]);
    //   hipMemcpy(&darray, &receiveBuffer_d[i], sizeof(transferPoints*), hipMemcpyDeviceToHost);
    //   hipMemcpy(receiveBuffer[i], darray, sizeof(transferPoints) * receivePoints[i], hipMemcpyDeviceToHost);
    // }

    if(myRank==0){
      cout<<"Deallocating memory and Destroying Communicators\n";
    }
    // 
    CUDACHECK(hipFree(splitPoint_d));
    // 
    NCCLCHECK(ncclCommDestroy(comm));
    MPI_Finalize();
    //
    auto duration = duration_cast<microseconds>(stop - start);
    cout << "Done with process "<<myRank<< ". Time Taken by was:" << duration.count() / 1000000.0 << endl;
}
