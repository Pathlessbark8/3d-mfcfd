// // #include <iostream>
// // #include <fstream>
// // #include <random>
// // #include "split_fluxes_mod.h"
// // #include <iomanip>
// // #include "octant_fluxes_mod.h"
// // #include "wall_flux_dGxneg_mod.h"
// // #include "wall_flux_dGyneg_mod.h"
// // #include "wall_flux_dGxpos_mod.h"
// // #include "wall_flux_dGypos_mod.h"
// // #include "wall_flux_dGzneg_mod.h"
// // #include "point_preprocessor_mod.h"
// // #include "compute_conserved_vector_mod.h"
// // #include "timestep_delt_mod.h"
// // #include "generate_connectivity_mod.h"
// // #include "implicit_aliasing_mod.h"
// // #include "flux_residual_mod.h"
// // #include "initial_conditions_mod.h"
// // #include "interior_flux_dGxneg_mod.h"
// // #include "interior_flux_dGxpos_mod.h"
// // #include "interior_flux_dGyneg_mod.h"
// // #include "interior_flux_dGypos_mod.h"
// // #include <chrono>
// // #include <hip/hip_runtime.h>
// // #include "fpi_solver_mod.h"
// // #include <nccl.h>
// // #include <mpi.h>

// // // #include "cuPrintf.cu"
// // using namespace std;
// // using namespace std::chrono;

// // #define MPICHECK(cmd) do {                          \
// //   int e = cmd;                                      \
// //   if( e != MPI_SUCCESS ) {                          \
// //     printf("Failed: MPI error %s:%d '%d'\n",        \
// //         __FILE__,__LINE__, e);   \
// //     exit(EXIT_FAILURE);                             \
// //   }                                                 \
// // } while(0)


// // #define CUDACHECK(cmd) do {                         \
// //   hipError_t e = cmd;                              \
// //   if( e != hipSuccess ) {                          \
// //     printf("Failed: Cuda error %s:%d '%s'\n",             \
// //         __FILE__,__LINE__,hipGetErrorString(e));   \
// //     exit(EXIT_FAILURE);                             \
// //   }                                                 \
// // } while(0)


// // #define NCCLCHECK(cmd) do {                         \
// //   ncclResult_t r = cmd;                             \
// //   if (r!= ncclSuccess) {                            \
// //     printf("Failed, NCCL error %s:%d '%s'\n",             \
// //         __FILE__,__LINE__,ncclGetErrorString(r));   \
// //     exit(EXIT_FAILURE);                             \
// //   }                                                 \
// // } while(0)


// // static uint64_t getHostHash(const char* string) {
// //   // Based on DJB2a, result = result * 33 ^ char
// //   uint64_t result = 5381;
// //   for (int c = 0; string[c] != '\0'; c++){
// //     result = ((result << 5) + result) ^ string[c];
// //   }
// //   return result;
// // }

// // int main(int argc, char* argv[])
// // {

// //     cout<<setprecision(13)<<scientific;

// //     read_input_point_data();
// //     initial_conditions();
// //     generate_split_stencils();
// //     //
// //     int myRank, nRanks, localRank = 0;
  
// //     //initializing MPI
// //     MPICHECK(MPI_Init(&argc, &argv));
// //     MPICHECK(MPI_Comm_rank(MPI_COMM_WORLD, &myRank));
// //     MPICHECK(MPI_Comm_size(MPI_COMM_WORLD, &nRanks));
// //     //
// //     fstream fin;
// //     fin.open("/home/anil/new_3d_code/3d-mfcfd/inputFiles/4DevicePartFile.dat",ios::in);
// //     fin>>numDevices;
// //     int counter;
// //     for(int i=0;i<max_points;i++){
// //         fin>>point.counter[i]>>partVector[i];
// //         if(partVector[i]/numDevices==myRank)
// //         {
// //           numberOfPointsPerDevice[partVector[i]%numDevices]++;
// //         }
// //     }
// //     fin.close();
// //     //
// //     cout<<numberOfPointsPerDevice<<" "<<numberOfPointsPerDevice[1]<<endl;
// //     //
// //     //
// //     //
// //     for(int i=0;i<numDevices;i++){
// //         splitPoint[i]=new splitPoints[numberOfPointsPerDevice[i]];
// //     }
// //     int *pointCounter=new int[numDevices];
// //     for(int i=0;i<numDevices;i++){
// //         pointCounter[i]=0;
// //     }
// //     cout<<"YO\n";
// //     for(int i=0;i<max_points;i++){
// //       if(partVector[i]/numDevices==myRank){
// //         assign(splitPoint[partVector[i]%numDevices][pointCounter[partVector[i]%numDevices]],i);
// //         pointCounter[partVector[i]%numDevices]++;
// //       }
// //     }
// //     //
// //     //
// //     comms=new ncclComm_t[numDevices];
// //     devs=new int[numDevices];
// //     for(int i=0;i<numDevices;i++){
// //         devs[i]=i;
// //     }
// //     cout<<"HI1\n";
// //     //
// //     splitPoints** splitPoint_d = (splitPoints**)malloc(numDevices * sizeof(splitPoints*));
// //     hipStream_t *s = (hipStream_t*)malloc(sizeof(hipStream_t)*numDevices);
// //     cout<<"HI2\n";
// //     cout<<numDevices<<endl;
// //     // //
// //     for (int i = 0; i < numDevices; ++i) {
// //         CUDACHECK(hipSetDevice(i));
// //         CUDACHECK(hipMalloc(splitPoint_d + i, numberOfPointsPerDevice[i] * sizeof(splitPoints)));
// //         CUDACHECK(hipStreamCreate(&s[i]));
// //     }
// //     for (int i = 0; i < numDevices; ++i) {
// //         CUDACHECK(hipSetDevice(i));
// //         CUDACHECK(hipMemcpyAsync(splitPoint_d[i], splitPoint[i], numberOfPointsPerDevice[i] * sizeof(splitPoints), hipMemcpyHostToDevice, s[i]));
// //     }
// //     //
// //     cout<<splitPoint[0][0].x<<" "<<splitPoint[1][1].x<<endl;
// //     cout<<"HI3\n";
// //     if (myRank == 0) ncclGetUniqueId(&id);
// //     MPICHECK(MPI_Bcast((void *)&id, sizeof(id), MPI_BYTE, 0, MPI_COMM_WORLD));
  
  
// //     // initializing NCCL, group API is required around ncclCommInitRank as it is
// //     // called across multiple GPUs in each thread/process
// //     // NCCLCHECK(ncclGroupStart());
// //     for (int i=0; i<numDevices; i++) {
// //        CUDACHECK(hipSetDevice(i));
// //        NCCLCHECK(ncclCommInitRank(comms+i, nRanks*numDevices, id, myRank*numDevices + i));
// //        cout<<"Rank "<<myRank<<" Device "<<i<<endl;
// //     }
// //     // NCCLCHECK(ncclGroupEnd());

// //     int count;
// //     ncclCommCount(comms[0], &count);
// //     cout<<myRank <<" Commcount "<<count<<endl; 

// //     ncclCommCuDevice(comms[0], &count);
// //     cout<<myRank <<" CuDevice "<<count<<endl;

// //     ncclCommUserRank(comms[1], &count);
// //     cout<<myRank <<" Comm user "<<count<<endl;
// //     for (int i=0; i<numDevices; i++)
// //       CUDACHECK(hipStreamSynchronize(s[i]));
// //     cout<<"HI4\n";
// //     // //
// //     fpi_solver_multi_nccl(splitPoint_d,s);
// //     // //
// //     cout<<"Copying memory back to Host\n";
// //     //
// //     for (int i = 0; i < numDevices; ++i) {
// //         CUDACHECK(hipSetDevice(i));
// //         CUDACHECK(hipMemcpyAsync(splitPoint[i], splitPoint_d[i], numberOfPointsPerDevice[i] * sizeof(splitPoints), hipMemcpyDeviceToHost, s[i]));
// //     }
// //     cout<<"Deallocating memory and Destroying Communicators\n";
// //     cout<<splitPoint[0][10].x<<endl;
// //     cout<<"HI7\n";
// //     //
// //     for (int i=0; i<numDevices; i++) {
// //       CUDACHECK(hipFree(splitPoint_d[i]));
// //     }
// //     //
// //     for (int i=0; i<numDevices; ++i){
// //         ncclCommDestroy(comms[i]);
// //     }
// //     MPI_Finalize();
// //     cout << "Done with process "<<myRank<<endl;
// // }

#include <iostream>
#include <fstream>
#include <random>
#include "split_fluxes_mod.h"
#include <iomanip>
#include "octant_fluxes_mod.h"
#include "wall_flux_dGxneg_mod.h"
#include "wall_flux_dGyneg_mod.h"
#include "wall_flux_dGxpos_mod.h"
#include "wall_flux_dGypos_mod.h"
#include "wall_flux_dGzneg_mod.h"
#include "point_preprocessor_mod.h"
#include "compute_conserved_vector_mod.h"
#include "timestep_delt_mod.h"
#include "generate_connectivity_mod.h"
#include "implicit_aliasing_mod.h"
#include "flux_residual_mod.h"
#include "initial_conditions_mod.h"
#include "interior_flux_dGxneg_mod.h"
#include "interior_flux_dGxpos_mod.h"
#include "interior_flux_dGyneg_mod.h"
#include "interior_flux_dGypos_mod.h"
#include <chrono>
#include <hip/hip_runtime.h>
#include "fpi_solver_mod.h"
#include <nccl.h>
#include <mpi.h>
#include <unistd.h>
#include <stdint.h>
#include <stdlib.h>
#include "mpi-ext.h"

// #include "cuPrintf.cu"
using namespace std;
using namespace std::chrono;



static uint64_t getHostHash(const char* string) {
  // Based on DJB2a, result = result * 33 ^ char
  uint64_t result = 5381;
  for (int c = 0; string[c] != '\0'; c++){
    result = ((result << 5) + result) ^ string[c];
  }
  return result;
}


static void getHostName(char* hostname, int maxlen) {
  gethostname(hostname, maxlen);
  for (int i=0; i< maxlen; i++) {
    if (hostname[i] == '.') {
        hostname[i] = '\0';
        return;
    }
  }
}

int main(int argc, char* argv[])
{

  printf("Compile time check:\n");
  #if defined(MPIX_CUDA_AWARE_SUPPORT) && MPIX_CUDA_AWARE_SUPPORT
      printf("This MPI library has CUDA-aware support.\n", MPIX_CUDA_AWARE_SUPPORT);
  #elif defined(MPIX_CUDA_AWARE_SUPPORT) && !MPIX_CUDA_AWARE_SUPPORT
      printf("This MPI library does not have CUDA-aware support.\n");
  #else
      printf("This MPI library cannot determine if there is CUDA-aware support.\n");
  #endif /* MPIX_CUDA_AWARE_SUPPORT */
   
      printf("Run time check:\n");
  #if defined(MPIX_CUDA_AWARE_SUPPORT)
      if (1 == MPIX_Query_cuda_support()) {
          printf("This MPI library has CUDA-aware support.\n");
      } else {
          printf("This MPI library does not have CUDA-aware support.\n");
      }
  #else /* !defined(MPIX_CUDA_AWARE_SUPPORT) */
      printf("This MPI library cannot determine if there is CUDA-aware support.\n");
  #endif /* MPIX_CUDA_AWARE_SUPPORT */


    //MAIN CODE BEGINS
    cout<<setprecision(13)<<scientific;

    read_input_point_data();
    initial_conditions();
    generate_split_stencils();
    //
    int myRank, nRanks, localRank = 0;
  
    //initializing MPI
    MPICHECK(MPI_Init(&argc, &argv));
    MPICHECK(MPI_Comm_rank(MPI_COMM_WORLD, &myRank));
    MPICHECK(MPI_Comm_size(MPI_COMM_WORLD, &nRanks));
    //
    cout<<nRanks<<endl;
    //
    // HASHING HOSTNAME TO GET LOCALRANKS
    uint64_t hostHashs[nRanks];
    char hostname[1024];
    getHostName(hostname, 1024);
    hostHashs[myRank] = getHostHash(hostname);
    MPICHECK(MPI_Allgather(MPI_IN_PLACE, 0, MPI_DATATYPE_NULL, hostHashs, sizeof(uint64_t), MPI_BYTE, MPI_COMM_WORLD));
    for (int p=0; p<nRanks; p++) {
      if (p == myRank) break;
      if (hostHashs[p] == hostHashs[myRank]) localRank++;
    }
  

    //READ POINTS FOR EACH DEVICE FROM FILE
    fstream fin;
    fin.open("/home/anil/new_3d_code/3d-mfcfd/inputFiles/filesFor"+to_string(nRanks)+"Devices/Device"+to_string(myRank)+".dat",ios::in);
    fin>>numDevices;
    fin>>local_points;
    int counter;
    localToGlobalIndex=new int [local_points];
    globalToGhostIndex=new int*[nRanks];
    for(int i=0;i<nRanks;i++)
    {
        globalToGhostIndex[i]=new int[max_points];
    }
    // partVector=new int [max_points];
    // ghostToGlobalIndex=new int *[nRanks];
    for(int i=0;i<local_points;i++){
        fin>>localToGlobalIndex[i];
        fin>>partVector[localToGlobalIndex[i]];
        globalToLocalIndex[localToGlobalIndex[i]]=i;
        // if(localToGlobalIndex[i]==442481){
        //   cout<<globalToLocalIndex[localToGlobalIndex[i]]<<" serhghtsdfghs \n";
        // }
        // if(partVector[i]==myRank)
        // {
        //   numberOfPointsPerDevice++;
        // }
        numberOfPointsPerDevice++;
    }
    MPICHECK(MPI_Allreduce(MPI_IN_PLACE, &partVector, max_points, MPI_INT, MPI_SUM, MPI_COMM_WORLD));
    // cout<<"CHECK : "<<globalToLocalIndex[432]<<endl;
    //
    ////ALLOCATING MEMORY FOR POINTS
    splitPoint=new splitPoints[numberOfPointsPerDevice];
    //
    //ASSIGNING POINTS FOR EACH DEVICE AND CALCULATING NATURE OF POINTS ON EACH PARTITION
    cout<<"Initialising....\n";
    for(int i=0;i<local_points;i++){
      // if(localToGlobalIndex[i]==442481){
      //   cout<<"Local Index is "<<i<<endl;
      // }
      assign(splitPoint[i],localToGlobalIndex[i],myRank);
      // if(localToGlobalIndex[i]==442481){
      //     cout<<"BIUHASDUCHUISHC "<<splitPoint[199977].globalIndex<<" "<<splitPoint[199977].x<<" "<<splitPoint[199977].y<<" "<<splitPoint[285282].z<<endl;
      // }
      findNatureOfLocalPoints(splitPoint[i]);
    }
    // cout<<"BIUHASDUC÷HUISHC "<<splitPoint[199977].globalIndex<<" "<<splitPoint[199977].x<<" "<<splitPoint[199977].y<<" "<<splitPoint[199977].z<<endl;
    allocateSizeForNatureOfLocalPoints();
    for(int i=0;i<local_points;i++){
      assignNatureOfLocalPoints(splitPoint[i],i);
    }

    // if(myRank==1){
    //   cout<<"Number Of Local Nbhs : "<<splitPoint[6081].numberOfGhostxposNbhs<<" "<<splitPoint[6081].numberOfLocalxposNbhs<<endl;
    // }

    // cout<<"Verification : "<<myRank<<" : "<<interiorPointsLocal<<" "<<wallPointsLocal<<" "<<outerPointsLocal<<" "<<interiorPointsLocal+wallPointsLocal+outerPointsLocal<<endl;
    //
    // cout<< " CHECK : "<< splitPoint[258].globalIndex<<endl;
    //
    sendBuffer=new transferPoints*[nRanks];
    int points_on_gpu_to_send_to;
    int total_points_to_send=0;
    for(int i=0;i<nRanks;i++){
        fin>>points_on_gpu_to_send_to;
        // ghostToGlobalIndex[i]=new int[points_on_gpu_to_send_to];
        total_points_to_send+=points_on_gpu_to_send_to;
        cout<<points_on_gpu_to_send_to<<endl;
        sendBuffer[i]=new transferPoints[points_on_gpu_to_send_to];
        // if(myRank==0)
        //   cout<<i<<" "<<points_on_gpu_to_send_to<<endl;
    }
    
    int currDevice=0;
    int *sendPoints=new int[nRanks];
    for(int i=0;i<nRanks;i++){
        sendPoints[i]=0;
    }
    cout<<endl;

    for(int i=0;i<total_points_to_send;++i){
        fin>>currDevice;
        fin>>setprecision(13)>>sendBuffer[currDevice][sendPoints[currDevice]].globalIndex>>sendBuffer[currDevice][sendPoints[currDevice]].x>>sendBuffer[currDevice][sendPoints[currDevice]].y>>sendBuffer[currDevice][sendPoints[currDevice]].z;
        int currIndex=globalToLocalIndex[sendBuffer[currDevice][sendPoints[currDevice]].globalIndex];
        splitPoint[currIndex].isGhost=true;
        splitPoint[currIndex].ghostIndex[splitPoint[currIndex].numberOfPartitionsToSendTo]=sendPoints[currDevice];
        splitPoint[currIndex].partitions[splitPoint[currIndex].numberOfPartitionsToSendTo]=currDevice;
        splitPoint[currIndex].numberOfPartitionsToSendTo++;
        // if(sendBuffer[currDevice][sendPoints[currDevice]].globalIndex==430334){
        //   cout<<"kjhedfjkehfjewhjf "<<sendPoints[currDevice];
        // }
        // if(sendBuffer[currDevice][sendPoints[currDevice]].globalIndex == 529172){
        //   cout<<"sioefhisdhfisdhflsdhflkshdlfksdlkf "<<sendPoints[currDevice]<<endl;
        // }
        // globalToGhostIndex[sendBuffer[currDevice][sendPoints[currDevice]].globalIndex]=sendPoints[currDevice];
        // if(sendBuffer[currDevice][sendPoints[currDevice]].globalIndex == 529172){
        //   cout<<"sioefhisdhfisdhflsdhflkshdlfksdlkf "<<globalToGhostIndex[sendBuffer[currDevice][sendPoints[currDevice]].globalIndex]<<endl;
        //   cout<<sendBuffer[currDevice][sendPoints[currDevice]].x<<" "<<sendBuffer[currDevice][sendPoints[currDevice]].y<<" "<<sendBuffer[currDevice][sendPoints[currDevice]].z<<endl;
        // }
        globalToGhostIndex[currDevice][sendBuffer[currDevice][sendPoints[currDevice]].globalIndex]=sendPoints[currDevice];
        // if(sendBuffer[currDevice][sendPoints[currDevice]].globalIndex == 333830){
        //   cout<<"sioefhisdhfisdhflsdhflkshdlfksdlkf "<<globalToGhostIndex[currDevice][sendBuffer[currDevice][sendPoints[currDevice]].globalIndex]<<endl;
        //   cout<<sendBuffer[currDevice][sendPoints[currDevice]].x<<" "<<sendBuffer[currDevice][sendPoints[currDevice]].y<<" "<<sendBuffer[currDevice][sendPoints[currDevice]].z<<endl;
        // }
        sendPoints[currDevice]++;
        
    }
    // if(myRank==0){
    //   cout<<"o;iwhfis;dhafidsahfisdhflshdfkdash "<<globalToGhostIndex[1][333830]<<endl;
    // }
    fin.close();
    // if(myRank==0){
    //   cout<<"o;iwhfis;dhafidsahfisdhflshdfkdash "<<globalToGhostIndex[1][333830]<<endl;
    // }
    
    cout<<"Number of Points in Process "<<myRank<<" are: "<<numberOfPointsPerDevice<<endl;
    //
    //
    cout<<"HI1\n";
    // //
    //Sharing Size across All Processes
    int *receivePoints=new int[nRanks];
    for(int i=0;i<nRanks;i++){
        receivePoints[i]=0;
    }
    for(int i=0;i<nRanks;i++){
      if(i!=myRank){
        MPI_Request request;
        MPICHECK(MPI_Isend(sendPoints+i , 1, MPI_INT, i, 0, MPI_COMM_WORLD,&request));
      }
    }
    for(int i=0;i<nRanks;i++){
      if(i!=myRank){
        // MPICHECK(MPI_Recv(receivePoints+i , 1, MPI_INT, MPI , 0, MPI_COMM_WORLD,MPI_STATUS_IGNORE));
        MPICHECK(MPI_Recv(receivePoints+i , 1, MPI_INT, i, 0, MPI_COMM_WORLD,MPI_STATUS_IGNORE));
      }
    }
    // cout<<myRank<<" "<<sendPoints[0]<<" "<<sendPoints[1]<<" "<<sendPoints[2]<<" "<<sendPoints[3]<<endl;
    // cout<<myRank<<" "<<receivePoints[0]<<" "<<receivePoints[1]<<" "<<receivePoints[2]<<" "<<receivePoints[3]<<endl;

    cout<<myRank<<" "<<sendPoints[0]<<" "<<sendPoints[1]<<endl;
    cout<<myRank<<" "<<receivePoints[0]<<" "<<receivePoints[1]<<endl;

    receiveBuffer=new transferPoints*[nRanks];
    for(int i=0;i<nRanks;i++){
      receiveBuffer[i]=new transferPoints[receivePoints[i]];
    }


    //CREATE MPI STRUCTURE TO TRANSFER POINTS DATA TO OTHER PROCESSES
    // const int nitems=5;
    // int          blocklengths[5] = {1,1,1,1,5};
    // MPI_Datatype types[5] = {MPI_INT, MPI_DOUBLE,MPI_DOUBLE,MPI_DOUBLE,MPI_DOUBLE};
    // MPI_Datatype mpi_transferPoints;
    // MPI_Aint     offsets[5];

    // offsets[0] = offsetof(transferPoints, counter);
    // offsets[1] = offsetof(transferPoints, x);
    // offsets[2] = offsetof(transferPoints, y);
    // offsets[3] = offsetof(transferPoints, z);
    // offsets[4] = offsetof(transferPoints, q);


    // MPI_Type_create_struct(nitems, blocklengths, offsets, types, &mpi_transferPoints);
    // MPI_Type_commit(&mpi_transferPoints);

    // for(int i=0;i<nRanks;i++){
    //   if(i!=myRank){
    //     MPICHECK(MPI_Send(sendBuffer[i] , sendPoints[i], MPI_BYTE, i, 0, MPI_COMM_WORLD));
    //     MPICHECK(MPI_Recv(receiveBuffer[myRank] , receivePoints[myRank], MPI_BYTE, myRank, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE));
    //   }
    // }

    // if(myRank==0){
    // cout<<myRank<<" "<<sendBuffer[1][0].x<<" "<<sendPoints[1]<<endl;
    // // cout<<myRank<<" "<<receiveBuffer[0]<<" "<<receivePoints[1]<<endl;
    // }
    cout<<"HI2\n";
    // cout<<partVector
    cout<<numDevices<<endl;
    // //
    int totalPointsToSend=0;
    for(int i=0;i<nRanks;i++){
      totalPointsToSend+=sendPoints[i];
    }
    cout<<"localRank: "<<localRank<<endl;
    cout<<myRank<<" "<<totalPointsToSend<<endl;


    splitPoints *splitPoint_d;
    int *globalToLocalIndex_temp;
    int **globalToGhostIndex_send,**globalToGhostIndex_receive;
    int **globalToGhostIndexSendPointer=(int**)malloc(sizeof(int*)*nRanks);
    int** globalToGhostIndexReceivePointer=(int**)malloc(sizeof(int*)*nRanks);

    int *partVector_d;

    transferPoints** sendBuffer_d,** receiveBuffer_d;
    transferPoints** sendPointer=(transferPoints**)malloc(sizeof(transferPoints*)*nRanks);
    transferPoints** receivePointer=(transferPoints**)malloc(sizeof(transferPoints*)*nRanks);
    CUDACHECK(hipSetDevice(localRank));

    //POINTER TO POINTER
    CUDACHECK(hipMalloc(&sendBuffer_d,sizeof(transferPoints*)*nRanks));
    for(int i = 0; i < nRanks; i++)
    {
      transferPoints *darray;
      hipMalloc(&darray, sizeof(transferPoints) * sendPoints[i]);
      hipMemcpy(darray,sendBuffer[i], sendPoints[i] * sizeof(transferPoints), hipMemcpyHostToDevice);
      hipMemcpy(&sendBuffer_d[i], &darray, sizeof(transferPoints*), hipMemcpyHostToDevice);
      memcpy(&sendPointer[i],&darray,sizeof(transferPoints*));
      // hipFree(darray);
      // hipMalloc(&sendBuffer_d[i],sizeof(transferPoints) * sendPoints[i]);
      // hipMemcpy(sendBuffer_d[i],sendBuffer[i], sendPoints[i] * sizeof(transferPoints), hipMemcpyHostToDevice);
    }

    // transferPoints** receiveBuffer_d;
    CUDACHECK(hipMalloc(&receiveBuffer_d,sizeof(transferPoints*)*nRanks));
    for(int i = 0; i < nRanks; i++)
    {
      transferPoints *darray;
      hipMalloc(&darray, sizeof(transferPoints) * receivePoints[i]);
      // cout<<"darray "<<&darray<<endl;
      // hipMemcpy(darray,sendBuffer[i], sendPoints[i] * sizeof(transferPoints), hipMemcpyHostToDevice);
      hipMemcpy(&receiveBuffer_d[i], &darray, sizeof(transferPoints*), hipMemcpyHostToDevice);
      memcpy(&receivePointer[i],&darray,sizeof(transferPoints*));
      // cout<<"darray "<<&receivePointer[i]<<endl;
      // hipFree(darray);
    }

    CUDACHECK(hipMalloc(&globalToGhostIndex_send, nRanks * sizeof(int*)));
    for(int i = 0; i < nRanks; i++)
    {
      int *darray;
      hipMalloc(&darray, sizeof(int) * max_points);
      hipMemcpy(darray,globalToGhostIndex[i], sizeof(int) * max_points, hipMemcpyHostToDevice);
      hipMemcpy(&globalToGhostIndex_send[i], &darray, sizeof(int*), hipMemcpyHostToDevice);
      memcpy(&globalToGhostIndexSendPointer[i],&darray,sizeof(int*));
    }
   

    CUDACHECK(hipMalloc(&globalToGhostIndex_receive, nRanks * sizeof(int*)));
    for(int i = 0; i < nRanks; i++)
    {
        int *darray;
        hipMalloc(&darray, sizeof(int) * max_points);
        hipMemcpy(&globalToGhostIndex_receive[i], &darray, sizeof(int*), hipMemcpyHostToDevice);
        memcpy(&globalToGhostIndexReceivePointer[i],&darray,sizeof(int*));
    }

    // CUDACHECK(hipStreamCreate(&s[0]));
        // CUDACHECK(hipSetDevice(i));
    // if(myRank==1){
    //   cout<<"o;iwhfis;dhafidsahfisdhflshdfkdash "<<globalToGhostIndex[529172]<<endl;
    // }
    CUDACHECK(hipMalloc(&splitPoint_d, numberOfPointsPerDevice * sizeof(splitPoints)));
    CUDACHECK(hipMemcpy(splitPoint_d, splitPoint, numberOfPointsPerDevice * sizeof(splitPoints), hipMemcpyHostToDevice));
    CUDACHECK(hipMalloc(&globalToLocalIndex_temp, max_points * sizeof(int)));
    CUDACHECK(hipMemcpy(globalToLocalIndex_temp, globalToLocalIndex, max_points * sizeof(int), hipMemcpyHostToDevice));
    CUDACHECK(hipMalloc(&partVector_d, max_points * sizeof(int)));
    CUDACHECK(hipMemcpy(partVector_d, &partVector, max_points * sizeof(int), hipMemcpyHostToDevice));
    
    // //
   
    // cout<<"STUCK "<<myRank<<endl;
    cout<<"HI3\n";
    if (myRank == 0) ncclGetUniqueId(&id);
    MPI_Barrier(MPI_COMM_WORLD);
    cout<<"Hiiiii\n";

    MPICHECK(MPI_Bcast((void *)&id, sizeof(id), MPI_BYTE, 0, MPI_COMM_WORLD));
    // //
    cout<<"Hiiiii\n";
    // //
    ncclComm_t comm;
    NCCLCHECK(ncclCommInitRank(&comm, nRanks, id, myRank));
    cout<<"Rank "<<myRank<<" Device "<<localRank<<endl;

    hipStream_t stream;
    // hipStreamCreateWithFlags(stream, hipStreamNonBlocking);
    // int count;
    // ncclCommCount(comm, &count);
    // cout<<myRank <<" Commcount "<<count<<endl; 

    // ncclCommCuDevice(comm, &count);
    // cout<<myRank <<" CuDevice "<<count<<endl;

    // ncclCommUserRank(comm, &count);
    // cout<<myRank <<" Comm user "<<count<<endl;
    // for (int i=0; i<numDevices; i++)
    //   CUDACHECK(hipDeviceSynchronize());
    cout<<"HI4\n";
    auto start = high_resolution_clock::now();
    // // //
    // cout<<"BIUHASDUCHUISHC "<<splitPoint[199977].globalIndex<<" "<<splitPoint[199977].x<<" "<<splitPoint[199977].y<<" "<<splitPoint[199977].z<<endl;

    fpi_solver_multi_nccl(splitPoint_d,localRank,sendBuffer_d,receiveBuffer_d,nRanks,myRank,sendPoints,receivePoints,comm,stream,sendPointer,receivePointer,globalToLocalIndex_temp,globalToGhostIndex_receive,globalToGhostIndexSendPointer,globalToGhostIndexReceivePointer,partVector_d);
    // //
    auto stop = high_resolution_clock::now();
    cout<<"Copying memory back to Host\n";
    //
    CUDACHECK(hipMemcpy(splitPoint, splitPoint_d, numberOfPointsPerDevice * sizeof(splitPoints), hipMemcpyDeviceToHost));

    
    // TO COPY BACK THE SEND BUFFER TO HOST (POINTER TO POINTER METHOD)
    transferPoints *darray;
    for (int i = 0; i <nRanks; i++){
      hipMalloc(&darray, sizeof(transferPoints) * sendPoints[i]);
      hipMemcpy(&darray, &sendBuffer_d[i], sizeof(transferPoints*), hipMemcpyDeviceToHost);
      hipMemcpy(sendBuffer[i], darray, sizeof(transferPoints) * sendPoints[i], hipMemcpyDeviceToHost);
      hipFree(darray);
    }

    // transferPoints *darray;
    for (int i = 0; i <nRanks; i++){
      hipMalloc(&darray, sizeof(transferPoints) * receivePoints[i]);
      hipMemcpy(&darray, &receiveBuffer_d[i], sizeof(transferPoints*), hipMemcpyDeviceToHost);
      hipMemcpy(receiveBuffer[i], darray, sizeof(transferPoints) * receivePoints[i], hipMemcpyDeviceToHost);
      // hipFree(darray);
    }

    // int *tempArray;
    // for (int i = 0; i <nRanks; i++){
    //   hipMalloc(&tempArray, sizeof(int) * max_points);
    //   hipMemcpy(&darray, &receiveBuffer_d[i], sizeof(transferPoints*), hipMemcpyDeviceToHost);
    //   hipMemcpy(receiveBuffer[i], darray, sizeof(transferPoints) * receivePoints[i], hipMemcpyDeviceToHost);
    //   // hipFree(darray);
    // }

    // if(myRank==0){
    // cout<<myRank<<" "<<sendBuffer[1][1].globalIndex<<" "<<sendBuffer[1][1].x<<endl;
    // for(int r=0;r<5;r++){
    //   cout<<"TEST "<<sendBuffer[1][1].q[r]<<endl;
    // }
    // }

    cout<<"Deallocating memory and Destroying Communicators\n";
    // if(myRank==1){
    // cout<<myRank<<" "<<receiveBuffer[0][1].globalIndex<<" "<<receiveBuffer[0][1].x<<endl;
    // for(int r=0;r<5;r++){
    //   cout<<"TEST recv buffer "<<receiveBuffer[0][1].q[r]<<endl;
    // }
    // }
    // cout<< " CHECK : "<< splitPoint[258].globalIndex<<endl;
    cout<<"HI7\n";
    // if(myRank==1){
    //   cout<<"kahfhdal"<<endl;
    //   cout<<myRank<<" "<<splitPoint[6081].globalIndex<<endl;
    //   // cout<<myRank<<" "<<splitPoint[6081].numberOfLocalxnegNbhs<<endl;
    //   cout<<myRank<<" "<<splitPoint[6081].numberOfGhostxnegNbhs<<endl;
    //   for(int r=0;r<5;r++){
    //     cout<<"TEST "<<splitPoint[6081].dq[0][r]<<endl;
    //   }
    // }
    // // //
    CUDACHECK(hipFree(splitPoint_d));
    // // //
    ncclCommDestroy(comm);
    MPI_Finalize();
    cout << "Done with process "<<myRank<<endl;
    
    auto duration = duration_cast<microseconds>(stop - start);
    cout << "Time Taken :" << duration.count() / 1000000.0 << endl;
}
