#include <iostream>
#include <fstream>
#include <random>
#include "split_fluxes_mod.h"
#include <iomanip>
#include "octant_fluxes_mod.h"
#include "wall_flux_dGxneg_mod.h"
#include "wall_flux_dGyneg_mod.h"
#include "wall_flux_dGxpos_mod.h"
#include "wall_flux_dGypos_mod.h"
#include "wall_flux_dGzneg_mod.h"
#include "point_preprocessor_mod.h"
#include "compute_conserved_vector_mod.h"
#include "timestep_delt_mod.h"
#include "generate_connectivity_mod.h"
#include "implicit_aliasing_mod.h"
#include "flux_residual_mod.h"
#include "initial_conditions_mod.h"
#include "interior_flux_dGxneg_mod.h"
#include "interior_flux_dGxpos_mod.h"
#include "interior_flux_dGyneg_mod.h"
#include "interior_flux_dGypos_mod.h"
#include <chrono>
#include <hip/hip_runtime.h>
#include "fpi_solver_mod.h"
using namespace std;
using namespace std::chrono;
int main()
{

    cout<<setprecision(13)<<scientific;

    read_input_point_data();
    initial_conditions();
    generate_split_stencils();
    //
    points *point_d;
    unsigned long long point_size = sizeof(point);
    hipStream_t stream;
    hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    //
    hipMalloc(&point_d, point_size);
    hipMemcpy(point_d, &point, point_size, hipMemcpyHostToDevice);
    // hipDeviceSynchronize();
    auto start = high_resolution_clock::now();
    cout << "Starting CUDA excecution\n";
    //
    cout << setprecision(13);
    fpi_solver_cuda(point_d,stream);
    //
    hipDeviceSynchronize();
    auto stop = high_resolution_clock::now();
    auto duration = duration_cast<microseconds>(stop - start);
    cout << "Time Taken :" << duration.count() / 1000000.0 << endl;
    //
    hipMemcpy(&point, point_d, point_size, hipMemcpyDeviceToHost);
    fstream fout;
    fout.open("output_prim.dat", ios::out);
    for(int i=0;i<max_points;++i){
        fout<<point.prim[0][i]<<" "<<point.prim[1][i]<<" "<<point.prim[2][i]<<" "<<point.prim[3][i]<<" "<<point.prim[4][i]<<endl;
    }
    fout.close();
    hipFree(point_d);
    //
    cout << "Done\n";
}