#include <iostream>
#include <fstream>
#include <random>
#include "split_fluxes_mod.h"
#include <iomanip>
#include "octant_fluxes_mod.h"
#include "wall_flux_dGxneg_mod.h"
#include "wall_flux_dGyneg_mod.h"
#include "wall_flux_dGxpos_mod.h"
#include "wall_flux_dGypos_mod.h"
#include "wall_flux_dGzneg_mod.h"
#include "point_preprocessor_mod.h"
#include "compute_conserved_vector_mod.h"
#include "timestep_delt_mod.h"
#include "generate_connectivity_mod.h"
#include "implicit_aliasing_mod.h"
#include "flux_residual_mod.h"
#include "initial_conditions_mod.h"
#include "interior_flux_dGxneg_mod.h"
#include "interior_flux_dGxpos_mod.h"
#include "interior_flux_dGyneg_mod.h"
#include "interior_flux_dGypos_mod.h"
#include <chrono>
#include <hip/hip_runtime.h>
#include "fpi_solver_mod.h"
#include <nccl.h>
#include <mpi.h>

// #include "cuPrintf.cu"
using namespace std;
using namespace std::chrono;

// #define MPICHECK(cmd) do {                          \
//     int e = cmd;                                      \
//     if( e != MPI_SUCCESS ) {                          \
//       printf("Failed: MPI error %s:%d '%d'\n",        \
//           __FILE__,__LINE__, e);   \
//       exit(EXIT_FAILURE);                             \
//     }                                                 \
//   } while(0)

#define CUDACHECK(cmd) do {                         \
    hipError_t err = cmd;                            \
    if (err != hipSuccess) {                         \
      printf("Failed: Cuda error %s:%d '%s'\n",       \
          __FILE__,__LINE__,hipGetErrorString(err)); \
      exit(EXIT_FAILURE);                             \
    }                                                 \
  } while(0)
  
  
  #define NCCLCHECK(cmd) do {                         \
    ncclResult_t res = cmd;                           \
    if (res != ncclSuccess) {                         \
      printf("Failed, NCCL error %s:%d '%s'\n",       \
          __FILE__,__LINE__,ncclGetErrorString(res)); \
      exit(EXIT_FAILURE);                             \
    }                                                 \
  } while(0)


int main()
{

    cout<<setprecision(13)<<scientific;

    read_input_point_data();
    initial_conditions();
    generate_split_stencils();
    //
    fstream fin;
    fin.open("/home/anil/new_3d_code/3d-mfcfd/inputFiles/partFile.dat",ios::in);
    fin>>numDevices;
    int counter;
    for(int i=0;i<max_points;i++){
        fin>>point.counter[i]>>partVector[i];
        numberOfPointsPerDevice[partVector[i]]++;
    }
    fin.close();
    //
    // cout<<numberOfPointsPerDevice[0]<<" "<<numberOfPointsPerDevice[1]<<" "<<numberOfPointsPerDevice[0]+numberOfPointsPerDevice[1]<<endl;
    //
    for(int i=0;i<numDevices;i++){
        splitPoint[i]=new splitPoints[numberOfPointsPerDevice[i]];
    }
    int *pointCounter=new int[numDevices];
    for(int i=0;i<numDevices;i++){
        pointCounter[i]=0;
    }
    for(int i=0;i<max_points;i++){
        assign(splitPoint[partVector[i]][pointCounter[partVector[i]]],i);
        pointCounter[partVector[i]]++;
    }
    //
    //
    // int myRank, nRanks;
    // MPI_Comm_rank(MPI_COMM_WORLD, &myRank);
    // MPI_Comm_size(MPI_COMM_WORLD, &nRanks);
    //
    comms=new ncclComm_t[numDevices];
    devs=new int[numDevices];
    for(int i=0;i<numDevices;i++){
        devs[i]=i;
    }
    cout<<"HI1\n";
    //
    splitPoints** splitPoint_d = (splitPoints**)malloc(numDevices * sizeof(splitPoints*));
    hipStream_t *s = (hipStream_t*)malloc(sizeof(hipStream_t)*numDevices);
    cout<<"HI2\n";
    cout<<numDevices<<endl;
    //
    // for(int i=0;i<numDevices;i++){
    //     CUDACHECK(hipStreamCreate(&s[i]));
    // }
    for (int i = 0; i < numDevices; ++i) {
        CUDACHECK(hipSetDevice(i));
        CUDACHECK(hipMalloc(splitPoint_d + i, numberOfPointsPerDevice[i] * sizeof(splitPoints)));
        CUDACHECK(hipStreamCreate(&s[i]));
        CUDACHECK(hipMemcpyAsync(splitPoint_d[i], splitPoint[i], numberOfPointsPerDevice[i] * sizeof(splitPoints), hipMemcpyHostToDevice, s[i]));
    }
    //
    cout<<splitPoint[0][0].x<<" "<<splitPoint[1][1].x<<endl;
    cout<<"HI3\n";
    for (int i = 0; i < numDevices; ++i) {
        CUDACHECK(hipSetDevice(i));
        CUDACHECK(hipStreamSynchronize(s[i]));
      }
    NCCLCHECK(ncclCommInitAll(comms, numDevices, devs));
    cout<<"HI4\n";
    // //
    fpi_solver_multi_nccl(splitPoint_d,s);
    // //
    cout<<"HI5\n";
    //
    // for (int i = 0; i < numDevices; ++i) {
    //     CUDACHECK(hipSetDevice(i));
    //     CUDACHECK(hipStreamSynchronize(s[i]));
    //   }

    for (int i = 0; i < numDevices; ++i) {
        CUDACHECK(hipSetDevice(i));
        CUDACHECK(hipMemcpyAsync(splitPoint[i], splitPoint_d[i], numberOfPointsPerDevice[i] * sizeof(splitPoints), hipMemcpyDeviceToHost, s[i]));
    }
    cout<<"HI6\n";
    // cout<<splitPoint[0][102].counter<<" "<<splitPoint[0][102].q[0]<<" "<<splitPoint[0][102].q[1]<<endl;
    cout<<splitPoint[0][0].x<<endl;
    cout<<"HI7\n";
    //
    // cudaPrintfDisplay(stdout, true);
    // cudaPrintfEnd();
    //
    for (int i=0; i<numDevices; ++i){
        ncclCommDestroy(comms[i]);
    }
    cout << "Done\n";
}

