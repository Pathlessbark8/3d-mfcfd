#include <iostream>
#include <fstream>
#include <random>
#include "split_fluxes_mod.h"
#include <iomanip>
#include "octant_fluxes_mod.h"
#include "wall_flux_dGxneg_mod.h"
#include "wall_flux_dGyneg_mod.h"
#include "wall_flux_dGxpos_mod.h"
#include "wall_flux_dGypos_mod.h"
#include "wall_flux_dGzneg_mod.h"
#include "point_preprocessor_mod.h"
#include "compute_conserved_vector_mod.h"
#include "timestep_delt_mod.h"
#include "generate_connectivity_mod.h"
#include "implicit_aliasing_mod.h"
#include "flux_residual_mod.h"
#include "initial_conditions_mod.h"
#include "interior_flux_dGxneg_mod.h"
#include "interior_flux_dGxpos_mod.h"
#include "interior_flux_dGyneg_mod.h"
#include "interior_flux_dGypos_mod.h"
#include <chrono>
#include <hip/hip_runtime.h>
#include "fpi_solver_mod.h"
#include <nccl.h>
#include <mpi.h>

// #include "cuPrintf.cu"
using namespace std;
using namespace std::chrono;

#define MPICHECK(cmd) do {                          \
    int e = cmd;                                      \
    if( e != MPI_SUCCESS ) {                          \
      printf("Failed: MPI error %s:%d '%d'\n",        \
          __FILE__,__LINE__, e);   \
      exit(EXIT_FAILURE);                             \
    }                                                 \
  } while(0)

#define CUDACHECK(cmd) do {                         \
    hipError_t err = cmd;                            \
    if (err != hipSuccess) {                         \
      printf("Failed: Cuda error %s:%d '%s'\n",       \
          __FILE__,__LINE__,hipGetErrorString(err)); \
      exit(EXIT_FAILURE);                             \
    }                                                 \
  } while(0)
  
  
  #define NCCLCHECK(cmd) do {                         \
    ncclResult_t res = cmd;                           \
    if (res != ncclSuccess) {                         \
      printf("Failed, NCCL error %s:%d '%s'\n",       \
          __FILE__,__LINE__,ncclGetErrorString(res)); \
      exit(EXIT_FAILURE);                             \
    }                                                 \
  } while(0)


int main(int argc, char* argv[])
{

    cout<<setprecision(13)<<scientific;

    read_input_point_data();
    initial_conditions();
    generate_split_stencils();
    //
    fstream fin;
    fin.open("/home/anil/new_3d_code/3d-mfcfd/inputFiles/partFile.dat",ios::in);
    fin>>numDevices;
    int counter;
    for(int i=0;i<max_points;i++){
        fin>>point.counter[i]>>partVector[i];
        numberOfPointsPerDevice[partVector[i]]++;
    }
    fin.close();
    //
    // cout<<numberOfPointsPerDevice[0]<<" "<<numberOfPointsPerDevice[1]<<" "<<numberOfPointsPerDevice[0]+numberOfPointsPerDevice[1]<<endl;
    //
    for(int i=0;i<numDevices;i++){
        splitPoint[i]=new splitPoints[numberOfPointsPerDevice[i]];
    }
    int *pointCounter=new int[numDevices];
    for(int i=0;i<numDevices;i++){
        pointCounter[i]=0;
    }
    for(int i=0;i<max_points;i++){
        assign(splitPoint[partVector[i]][pointCounter[partVector[i]]],i);
        pointCounter[partVector[i]]++;
    }
    //
    //
    int myRank, nRanks, localRank = 0;
  
  
    //initializing MPI
    MPICHECK(MPI_Init(&argc, &argv));
    MPICHECK(MPI_Comm_rank(MPI_COMM_WORLD, &myRank));
    MPICHECK(MPI_Comm_size(MPI_COMM_WORLD, &nRanks));
    //
    comms=new ncclComm_t[numDevices];
    devs=new int[numDevices];
    for(int i=0;i<numDevices;i++){
        devs[i]=i;
    }
    cout<<"HI1\n";
    //
    splitPoints** splitPoint_d = (splitPoints**)malloc(numDevices * sizeof(splitPoints*));
    hipStream_t *s = (hipStream_t*)malloc(sizeof(hipStream_t)*numDevices);
    cout<<"HI2\n";
    cout<<numDevices<<endl;
    //
    for (int i = 0; i < numDevices; ++i) {
        CUDACHECK(hipSetDevice(i));
        CUDACHECK(hipMalloc(splitPoint_d + i, numberOfPointsPerDevice[i] * sizeof(splitPoints)));
        CUDACHECK(hipStreamCreate(&s[i]));
        CUDACHECK(hipMemcpyAsync(splitPoint_d[i], splitPoint[i], numberOfPointsPerDevice[i] * sizeof(splitPoints), hipMemcpyHostToDevice, s[i]));
    }
    //
    cout<<splitPoint[0][0].x<<" "<<splitPoint[1][1].x<<endl;
    cout<<"HI3\n";
    if (myRank == 0) ncclGetUniqueId(&id);
    MPICHECK(MPI_Bcast((void *)&id, sizeof(id), MPI_BYTE, 0, MPI_COMM_WORLD));
  
  
    // initializing NCCL, group API is required around ncclCommInitRank as it is
    // called across multiple GPUs in each thread/process
    NCCLCHECK(ncclGroupStart());
    for (int i=0; i<numDevices; i++) {
       CUDACHECK(hipSetDevice(localRank*numDevices + i));
       NCCLCHECK(ncclCommInitRank(comms+i, nRanks*numDevices, id, myRank*numDevices + i));
    }
    NCCLCHECK(ncclGroupEnd());

    for (int i=0; i<numDevices; i++)
      CUDACHECK(hipStreamSynchronize(s[i]));
    cout<<"HI4\n";
    // //
    fpi_solver_multi_nccl(splitPoint_d,s);
    // //
    cout<<"HI5\n";
    //
    for (int i = 0; i < numDevices; ++i) {
        CUDACHECK(hipSetDevice(i));
        CUDACHECK(hipMemcpyAsync(splitPoint[i], splitPoint_d[i], numberOfPointsPerDevice[i] * sizeof(splitPoints), hipMemcpyDeviceToHost, s[i]));
    }
    cout<<"HI6\n";
    cout<<splitPoint[0][0].x<<endl;
    cout<<"HI7\n";
    //
    for (int i=0; i<numDevices; i++) {
      CUDACHECK(hipFree(splitPoint_d[i]));
    }
    //
    for (int i=0; i<numDevices; ++i){
        ncclCommDestroy(comms[i]);
    }
    cout << "Done with process "<<myRank<<endl;
}

// #include <stdio.h>
// #include "hip/hip_runtime.h"
// #include "nccl.h"
// #include "mpi.h"
// #include <unistd.h>
// #include <stdint.h>
// #include <iostream>


// #define MPICHECK(cmd) do {                          \
//   int e = cmd;                                      \
//   if( e != MPI_SUCCESS ) {                          \
//     printf("Failed: MPI error %s:%d '%d'\n",        \
//         __FILE__,__LINE__, e);   \
//     exit(EXIT_FAILURE);                             \
//   }                                                 \
// } while(0)


// #define CUDACHECK(cmd) do {                         \
//   hipError_t e = cmd;                              \
//   if( e != hipSuccess ) {                          \
//     printf("Failed: Cuda error %s:%d '%s'\n",             \
//         __FILE__,__LINE__,hipGetErrorString(e));   \
//     exit(EXIT_FAILURE);                             \
//   }                                                 \
// } while(0)


// #define NCCLCHECK(cmd) do {                         \
//   ncclResult_t r = cmd;                             \
//   if (r!= ncclSuccess) {                            \
//     printf("Failed, NCCL error %s:%d '%s'\n",             \
//         __FILE__,__LINE__,ncclGetErrorString(r));   \
//     exit(EXIT_FAILURE);                             \
//   }                                                 \
// } while(0)


// static uint64_t getHostHash(const char* string) {
//   // Based on DJB2a, result = result * 33 ^ char
//   uint64_t result = 5381;
//   for (int c = 0; string[c] != '\0'; c++){
//     result = ((result << 5) + result) ^ string[c];
//   }
//   return result;
// }


// static void getHostName(char* hostname, int maxlen) {
//   gethostname(hostname, maxlen);
//   for (int i=0; i< maxlen; i++) {
//     if (hostname[i] == '.') {
//         hostname[i] = '\0';
//         return;
//     }
//   }
// }


// int main(int argc, char* argv[])
// {
//   int size = 32*1024*1024;


//   int myRank, nRanks, localRank = 0;


//   //initializing MPI
//   MPICHECK(MPI_Init(&argc, &argv));
//   MPICHECK(MPI_Comm_rank(MPI_COMM_WORLD, &myRank));
//   MPICHECK(MPI_Comm_size(MPI_COMM_WORLD, &nRanks));

//   printf("Rank %d out of %d processors\n", myRank, nRanks);

//   //calculating localRank which is used in selecting a GPU
//   // uint64_t hostHashs[nRanks];
//   // char hostname[1024];
//   // getHostName(hostname, 1024);
//   // hostHashs[myRank] = getHostHash(hostname);
//   // std::cout<<"HostHash: "<<hostHashs[myRank]<<std::endl;
//   // MPICHECK(MPI_Allgather(MPI_IN_PLACE, 0, MPI_DATATYPE_NULL, hostHashs, sizeof(uint64_t), MPI_BYTE, MPI_COMM_WORLD));
//   // for (int i=0; i<nRanks; i++) {
//   //   // std::cout<<hostHashs[i]<<std::endl;
//   // }
//   // for (int p=0; p<nRanks; p++) {
//   //   // std::cout<<"p: "<<p<<std::endl;
//   //    if (p == myRank) break;
//   //    std::cout<<"p: "<<p<<" HostHash: "<<hostHashs[p]<<" Rank "<<myRank<<std::endl;
//   //    if (hostHashs[p] == hostHashs[myRank]) {
//   //     localRank++;
//   //     std::cout<<localRank<<std::endl;
//   //    }
//   // }


//   //each process is using two GPUs
//   int nDev = 2;


//   float** sendbuff = (float**)malloc(nDev * sizeof(float*));
//   float** recvbuff = (float**)malloc(nDev * sizeof(float*));
//   hipStream_t* s = (hipStream_t*)malloc(sizeof(hipStream_t)*nDev);


//   //picking GPUs based on localRank
//   for (int i = 0; i < nDev; ++i) {
//     // std::cout<<myRank<<" "<<localRank<<" "<<i<<std::endl;
//     CUDACHECK(hipSetDevice(localRank*nDev + i));
//     CUDACHECK(hipMalloc(sendbuff + i, size * sizeof(float)));
//     CUDACHECK(hipMalloc(recvbuff + i, size * sizeof(float)));
//     CUDACHECK(hipMemset(sendbuff[i], 1, size * sizeof(float)));
//     CUDACHECK(hipMemset(recvbuff[i], 0, size * sizeof(float)));
//     CUDACHECK(hipStreamCreate(s+i));
//   }


//   ncclUniqueId id;
//   ncclComm_t comms[nDev];

//   //generating NCCL unique ID at one process and broadcasting it to all
//   if (myRank == 0) ncclGetUniqueId(&id);
//   MPICHECK(MPI_Bcast((void *)&id, sizeof(id), MPI_BYTE, 0, MPI_COMM_WORLD));


//   //initializing NCCL, group API is required around ncclCommInitRank as it is
//   //called across multiple GPUs in each thread/process
//   NCCLCHECK(ncclGroupStart());
//   for (int i=0; i<nDev; i++) {
//      CUDACHECK(hipSetDevice(localRank*nDev + i));
//      NCCLCHECK(ncclCommInitRank(comms+i, nRanks*nDev, id, myRank*nDev + i));
//   }
//   NCCLCHECK(ncclGroupEnd());


//   //calling NCCL communication API. Group API is required when using
//   //multiple devices per thread/process
//   NCCLCHECK(ncclGroupStart());
//   for (int i=0; i<nDev; i++)
//      NCCLCHECK(ncclAllReduce((const void*)sendbuff[i], (void*)recvbuff[i], size, ncclFloat, ncclSum,
//            comms[i], s[i]));
//   NCCLCHECK(ncclGroupEnd());


//   //synchronizing on CUDA stream to complete NCCL communication
//   for (int i=0; i<nDev; i++)
//       CUDACHECK(hipStreamSynchronize(s[i]));


//   //freeing device memory
//   for (int i=0; i<nDev; i++) {
//      CUDACHECK(hipFree(sendbuff[i]));
//      CUDACHECK(hipFree(recvbuff[i]));
//   }


//   //finalizing NCCL
//   for (int i=0; i<nDev; i++) {
//      ncclCommDestroy(comms[i]);
//   }


//   //finalizing MPI
//   MPICHECK(MPI_Finalize());


//   printf("[MPI Rank %d] Success \n", myRank);
//   return 0;
// }