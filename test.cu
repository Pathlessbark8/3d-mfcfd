// // #include <iostream>
// // #include <fstream>
// // #include <random>
// // #include "split_fluxes_mod.h"
// // #include <iomanip>
// // #include "octant_fluxes_mod.h"
// // #include "wall_flux_dGxneg_mod.h"
// // #include "wall_flux_dGyneg_mod.h"
// // #include "wall_flux_dGxpos_mod.h"
// // #include "wall_flux_dGypos_mod.h"
// // #include "wall_flux_dGzneg_mod.h"
// // #include "point_preprocessor_mod.h"
// // #include "compute_conserved_vector_mod.h"
// // #include "timestep_delt_mod.h"
// // #include "generate_connectivity_mod.h"
// // #include "implicit_aliasing_mod.h"
// // #include "flux_residual_mod.h"
// // #include "initial_conditions_mod.h"
// // #include "interior_flux_dGxneg_mod.h"
// // #include "interior_flux_dGxpos_mod.h"
// // #include "interior_flux_dGyneg_mod.h"
// // #include "interior_flux_dGypos_mod.h"
// // #include <chrono>
// // #include <hip/hip_runtime.h>
// // #include "fpi_solver_mod.h"
// // #include <nccl.h>
// // #include <mpi.h>

// // // #include "cuPrintf.cu"
// // using namespace std;
// // using namespace std::chrono;

// // #define MPICHECK(cmd) do {                          \
// //   int e = cmd;                                      \
// //   if( e != MPI_SUCCESS ) {                          \
// //     printf("Failed: MPI error %s:%d '%d'\n",        \
// //         __FILE__,__LINE__, e);   \
// //     exit(EXIT_FAILURE);                             \
// //   }                                                 \
// // } while(0)


// // #define CUDACHECK(cmd) do {                         \
// //   hipError_t e = cmd;                              \
// //   if( e != hipSuccess ) {                          \
// //     printf("Failed: Cuda error %s:%d '%s'\n",             \
// //         __FILE__,__LINE__,hipGetErrorString(e));   \
// //     exit(EXIT_FAILURE);                             \
// //   }                                                 \
// // } while(0)


// // #define NCCLCHECK(cmd) do {                         \
// //   ncclResult_t r = cmd;                             \
// //   if (r!= ncclSuccess) {                            \
// //     printf("Failed, NCCL error %s:%d '%s'\n",             \
// //         __FILE__,__LINE__,ncclGetErrorString(r));   \
// //     exit(EXIT_FAILURE);                             \
// //   }                                                 \
// // } while(0)


// // static uint64_t getHostHash(const char* string) {
// //   // Based on DJB2a, result = result * 33 ^ char
// //   uint64_t result = 5381;
// //   for (int c = 0; string[c] != '\0'; c++){
// //     result = ((result << 5) + result) ^ string[c];
// //   }
// //   return result;
// // }

// // int main(int argc, char* argv[])
// // {

// //     cout<<setprecision(13)<<scientific;

// //     read_input_point_data();
// //     initial_conditions();
// //     generate_split_stencils();
// //     //
// //     int myRank, nRanks, localRank = 0;
  
// //     //initializing MPI
// //     MPICHECK(MPI_Init(&argc, &argv));
// //     MPICHECK(MPI_Comm_rank(MPI_COMM_WORLD, &myRank));
// //     MPICHECK(MPI_Comm_size(MPI_COMM_WORLD, &nRanks));
// //     //
// //     fstream fin;
// //     fin.open("/home/anil/new_3d_code/3d-mfcfd/inputFiles/4DevicePartFile.dat",ios::in);
// //     fin>>numDevices;
// //     int counter;
// //     for(int i=0;i<max_points;i++){
// //         fin>>point.counter[i]>>partVector[i];
// //         if(partVector[i]/numDevices==myRank)
// //         {
// //           numberOfPointsPerDevice[partVector[i]%numDevices]++;
// //         }
// //     }
// //     fin.close();
// //     //
// //     cout<<numberOfPointsPerDevice<<" "<<numberOfPointsPerDevice[1]<<endl;
// //     //
// //     //
// //     //
// //     for(int i=0;i<numDevices;i++){
// //         splitPoint[i]=new splitPoints[numberOfPointsPerDevice[i]];
// //     }
// //     int *pointCounter=new int[numDevices];
// //     for(int i=0;i<numDevices;i++){
// //         pointCounter[i]=0;
// //     }
// //     cout<<"YO\n";
// //     for(int i=0;i<max_points;i++){
// //       if(partVector[i]/numDevices==myRank){
// //         assign(splitPoint[partVector[i]%numDevices][pointCounter[partVector[i]%numDevices]],i);
// //         pointCounter[partVector[i]%numDevices]++;
// //       }
// //     }
// //     //
// //     //
// //     comms=new ncclComm_t[numDevices];
// //     devs=new int[numDevices];
// //     for(int i=0;i<numDevices;i++){
// //         devs[i]=i;
// //     }
// //     cout<<"HI1\n";
// //     //
// //     splitPoints** splitPoint_d = (splitPoints**)malloc(numDevices * sizeof(splitPoints*));
// //     hipStream_t *s = (hipStream_t*)malloc(sizeof(hipStream_t)*numDevices);
// //     cout<<"HI2\n";
// //     cout<<numDevices<<endl;
// //     // //
// //     for (int i = 0; i < numDevices; ++i) {
// //         CUDACHECK(hipSetDevice(i));
// //         CUDACHECK(hipMalloc(splitPoint_d + i, numberOfPointsPerDevice[i] * sizeof(splitPoints)));
// //         CUDACHECK(hipStreamCreate(&s[i]));
// //     }
// //     for (int i = 0; i < numDevices; ++i) {
// //         CUDACHECK(hipSetDevice(i));
// //         CUDACHECK(hipMemcpyAsync(splitPoint_d[i], splitPoint[i], numberOfPointsPerDevice[i] * sizeof(splitPoints), hipMemcpyHostToDevice, s[i]));
// //     }
// //     //
// //     cout<<splitPoint[0][0].x<<" "<<splitPoint[1][1].x<<endl;
// //     cout<<"HI3\n";
// //     if (myRank == 0) ncclGetUniqueId(&id);
// //     MPICHECK(MPI_Bcast((void *)&id, sizeof(id), MPI_BYTE, 0, MPI_COMM_WORLD));
  
  
// //     // initializing NCCL, group API is required around ncclCommInitRank as it is
// //     // called across multiple GPUs in each thread/process
// //     // NCCLCHECK(ncclGroupStart());
// //     for (int i=0; i<numDevices; i++) {
// //        CUDACHECK(hipSetDevice(i));
// //        NCCLCHECK(ncclCommInitRank(comms+i, nRanks*numDevices, id, myRank*numDevices + i));
// //        cout<<"Rank "<<myRank<<" Device "<<i<<endl;
// //     }
// //     // NCCLCHECK(ncclGroupEnd());

// //     int count;
// //     ncclCommCount(comms[0], &count);
// //     cout<<myRank <<" Commcount "<<count<<endl; 

// //     ncclCommCuDevice(comms[0], &count);
// //     cout<<myRank <<" CuDevice "<<count<<endl;

// //     ncclCommUserRank(comms[1], &count);
// //     cout<<myRank <<" Comm user "<<count<<endl;
// //     for (int i=0; i<numDevices; i++)
// //       CUDACHECK(hipStreamSynchronize(s[i]));
// //     cout<<"HI4\n";
// //     // //
// //     fpi_solver_multi_nccl(splitPoint_d,s);
// //     // //
// //     cout<<"Copying memory back to Host\n";
// //     //
// //     for (int i = 0; i < numDevices; ++i) {
// //         CUDACHECK(hipSetDevice(i));
// //         CUDACHECK(hipMemcpyAsync(splitPoint[i], splitPoint_d[i], numberOfPointsPerDevice[i] * sizeof(splitPoints), hipMemcpyDeviceToHost, s[i]));
// //     }
// //     cout<<"Deallocating memory and Destroying Communicators\n";
// //     cout<<splitPoint[0][10].x<<endl;
// //     cout<<"HI7\n";
// //     //
// //     for (int i=0; i<numDevices; i++) {
// //       CUDACHECK(hipFree(splitPoint_d[i]));
// //     }
// //     //
// //     for (int i=0; i<numDevices; ++i){
// //         ncclCommDestroy(comms[i]);
// //     }
// //     MPI_Finalize();
// //     cout << "Done with process "<<myRank<<endl;
// // }

#include <iostream>
#include <fstream>
#include <random>
#include "split_fluxes_mod.h"
#include <iomanip>
#include "octant_fluxes_mod.h"
#include "wall_flux_dGxneg_mod.h"
#include "wall_flux_dGyneg_mod.h"
#include "wall_flux_dGxpos_mod.h"
#include "wall_flux_dGypos_mod.h"
#include "wall_flux_dGzneg_mod.h"
#include "point_preprocessor_mod.h"
#include "compute_conserved_vector_mod.h"
#include "timestep_delt_mod.h"
#include "generate_connectivity_mod.h"
#include "implicit_aliasing_mod.h"
#include "flux_residual_mod.h"
#include "initial_conditions_mod.h"
#include "interior_flux_dGxneg_mod.h"
#include "interior_flux_dGxpos_mod.h"
#include "interior_flux_dGyneg_mod.h"
#include "interior_flux_dGypos_mod.h"
#include <chrono>
#include <hip/hip_runtime.h>
#include "fpi_solver_mod.h"
#include <nccl.h>
#include <mpi.h>
#include <unistd.h>
#include <stdint.h>
#include <stdlib.h>
#include "mpi-ext.h"

// #include "cuPrintf.cu"
using namespace std;
using namespace std::chrono;



static uint64_t getHostHash(const char* string) {
  // Based on DJB2a, result = result * 33 ^ char
  uint64_t result = 5381;
  for (int c = 0; string[c] != '\0'; c++){
    result = ((result << 5) + result) ^ string[c];
  }
  return result;
}


static void getHostName(char* hostname, int maxlen) {
  gethostname(hostname, maxlen);
  for (int i=0; i< maxlen; i++) {
    if (hostname[i] == '.') {
        hostname[i] = '\0';
        return;
    }
  }
}

int main(int argc, char* argv[])
{

    //MAIN CODE BEGINS
    cout<<setprecision(13)<<scientific;

    read_input_point_data();
    initial_conditions();
    generate_split_stencils();
    //
    int myRank, nRanks, localRank = 0;
  
    //initializing MPI
    MPICHECK(MPI_Init(&argc, &argv));
    MPICHECK(MPI_Comm_rank(MPI_COMM_WORLD, &myRank));
    MPICHECK(MPI_Comm_size(MPI_COMM_WORLD, &nRanks));
    //

    //CHECK IF MPI LIBRARY HAD CUDA SUPPORT
    if(myRank==0){
      printf("Compile time check:\n");
      #if defined(MPIX_CUDA_AWARE_SUPPORT) && MPIX_CUDA_AWARE_SUPPORT
          printf("This MPI library has CUDA-aware support.\n", MPIX_CUDA_AWARE_SUPPORT);
      #elif defined(MPIX_CUDA_AWARE_SUPPORT) && !MPIX_CUDA_AWARE_SUPPORT
          printf("This MPI library does not have CUDA-aware support.\n");
      #else
          printf("This MPI library cannot determine if there is CUDA-aware support.\n");
      #endif /* MPIX_CUDA_AWARE_SUPPORT */
      
          printf("Run time check:\n");
      #if defined(MPIX_CUDA_AWARE_SUPPORT)
          if (1 == MPIX_Query_cuda_support()) {
              printf("This MPI library has CUDA-aware support.\n");
          } else {
              printf("This MPI library does not have CUDA-aware support.\n");
          }
      #else /* !defined(MPIX_CUDA_AWARE_SUPPORT) */
          printf("This MPI library cannot determine if there is CUDA-aware support.\n");
      #endif /* MPIX_CUDA_AWARE_SUPPORT */

      cout<<"Total Number of Proccess are "<<nRanks<<endl;
    }
    //
    // HASHING HOSTNAME TO GET LOCALRANKS
    uint64_t hostHashs[nRanks];
    char hostname[1024];
    getHostName(hostname, 1024);
    hostHashs[myRank] = getHostHash(hostname);
    MPICHECK(MPI_Allgather(MPI_IN_PLACE, 0, MPI_DATATYPE_NULL, hostHashs, sizeof(uint64_t), MPI_BYTE, MPI_COMM_WORLD));
    for (int p=0; p<nRanks; p++) {
      if (p == myRank) break;
      if (hostHashs[p] == hostHashs[myRank]) localRank++;
    }
  
    if(myRank==0){
      cout<<"Reading from File\n";
    }
    //READ POINTS FOR EACH DEVICE FROM FILE
    fstream fin;
    fin.open("/home/anil/new_3d_code/3d-mfcfd/inputFiles/filesFor"+to_string(nRanks)+"Devices/Device"+to_string(myRank)+".dat",ios::in);
    fin>>numDevices;
    fin>>local_points;
    int counter;
    localToGlobalIndex=new int [local_points];
    globalToGhostIndex=new int*[nRanks];
    for(int i=0;i<nRanks;i++)
    {
        globalToGhostIndex[i]=new int[max_points];
    }

    for(int i=0;i<local_points;i++){
        fin>>localToGlobalIndex[i];
        fin>>partVector[localToGlobalIndex[i]];
        globalToLocalIndex[localToGlobalIndex[i]]=i;
        numberOfPointsPerDevice++;
    }

    //Share Partition Value across Proccesses
    MPICHECK(MPI_Allreduce(MPI_IN_PLACE, &partVector, max_points, MPI_INT, MPI_SUM, MPI_COMM_WORLD));    

    //ALLOCATING MEMORY FOR POINTS
    splitPoint=new splitPoints[numberOfPointsPerDevice];

    if(myRank==0){
      cout<<"Determining Nature of Points\n";
    }
    //ASSIGNING POINTS FOR EACH DEVICE AND CALCULATING NATURE OF POINTS ON EACH PARTITION
    for(int i=0;i<local_points;i++){
      assign(splitPoint[i],localToGlobalIndex[i],myRank);
      findNatureOfLocalPoints(splitPoint[i]);
    }
    allocateSizeForNatureOfLocalPoints();
    for(int i=0;i<local_points;i++){
      assignNatureOfLocalPoints(splitPoint[i],i);
    }

    //Initialising the Send Buffer
    sendBuffer=new transferPoints*[nRanks];
    int points_on_gpu_to_send_to;
    int total_points_to_send=0;
    for(int i=0;i<nRanks;i++){
        fin>>points_on_gpu_to_send_to;
        total_points_to_send+=points_on_gpu_to_send_to;
        sendBuffer[i]=new transferPoints[points_on_gpu_to_send_to];
    }
    
    int currDevice=0;
    int *sendPoints=new int[nRanks];
    for(int i=0;i<nRanks;i++){
        sendPoints[i]=0;
    }

    //Populating the Send Buffer with Data
    for(int i=0;i<total_points_to_send;++i){
        fin>>currDevice;
        fin>>setprecision(13)>>sendBuffer[currDevice][sendPoints[currDevice]].globalIndex>>sendBuffer[currDevice][sendPoints[currDevice]].x>>sendBuffer[currDevice][sendPoints[currDevice]].y>>sendBuffer[currDevice][sendPoints[currDevice]].z;
        int currIndex=globalToLocalIndex[sendBuffer[currDevice][sendPoints[currDevice]].globalIndex];
        splitPoint[currIndex].isGhost=true;
        splitPoint[currIndex].ghostIndex[splitPoint[currIndex].numberOfPartitionsToSendTo]=sendPoints[currDevice];
        splitPoint[currIndex].partitions[splitPoint[currIndex].numberOfPartitionsToSendTo]=currDevice;
        splitPoint[currIndex].numberOfPartitionsToSendTo++;
        globalToGhostIndex[currDevice][sendBuffer[currDevice][sendPoints[currDevice]].globalIndex]=sendPoints[currDevice];
        sendPoints[currDevice]++;   
    }
    fin.close();
    
    cout<<"Number of Points in Process "<<myRank<<" are: "<<numberOfPointsPerDevice<<endl;

    //Sharing Size across All Processes
    int *receivePoints=new int[nRanks];
    for(int i=0;i<nRanks;i++){
        receivePoints[i]=0;
    }
    for(int i=0;i<nRanks;i++){
      if(i!=myRank){
        MPI_Request request;
        MPICHECK(MPI_Isend(sendPoints+i , 1, MPI_INT, i, 0, MPI_COMM_WORLD,&request));
      }
    }
    for(int i=0;i<nRanks;i++){
      if(i!=myRank){
        // MPICHECK(MPI_Recv(receivePoints+i , 1, MPI_INT, MPI , 0, MPI_COMM_WORLD,MPI_STATUS_IGNORE));
        MPICHECK(MPI_Recv(receivePoints+i , 1, MPI_INT, i, 0, MPI_COMM_WORLD,MPI_STATUS_IGNORE));
      }
    }

    receiveBuffer=new transferPoints*[nRanks];
    for(int i=0;i<nRanks;i++){
      receiveBuffer[i]=new transferPoints[receivePoints[i]];
    }


    //CREATE MPI STRUCTURE TO TRANSFER POINTS DATA TO OTHER PROCESSES
    // const int nitems=5;
    // int          blocklengths[5] = {1,1,1,1,5};
    // MPI_Datatype types[5] = {MPI_INT, MPI_DOUBLE,MPI_DOUBLE,MPI_DOUBLE,MPI_DOUBLE};
    // MPI_Datatype mpi_transferPoints;
    // MPI_Aint     offsets[5];

    // offsets[0] = offsetof(transferPoints, counter);
    // offsets[1] = offsetof(transferPoints, x);
    // offsets[2] = offsetof(transferPoints, y);
    // offsets[3] = offsetof(transferPoints, z);
    // offsets[4] = offsetof(transferPoints, q);


    // MPI_Type_create_struct(nitems, blocklengths, offsets, types, &mpi_transferPoints);
    // MPI_Type_commit(&mpi_transferPoints);
    // //

    int totalPointsToSend=0;
    for(int i=0;i<nRanks;i++){
      totalPointsToSend+=sendPoints[i];
    }

    //Initialising and transfering memory to device Pointers
    splitPoints *splitPoint_d;

    int *globalToLocalIndex_temp;
    int **globalToGhostIndex_send,**globalToGhostIndex_receive;
    int **globalToGhostIndexSendPointer=(int**)malloc(sizeof(int*)*nRanks);
    int** globalToGhostIndexReceivePointer=(int**)malloc(sizeof(int*)*nRanks);
    int *partVector_d;

    transferPoints** sendBuffer_d,** receiveBuffer_d;
    transferPoints** sendPointer=(transferPoints**)malloc(sizeof(transferPoints*)*nRanks);
    transferPoints** receivePointer=(transferPoints**)malloc(sizeof(transferPoints*)*nRanks);
    CUDACHECK(hipSetDevice(localRank));

    //POINTER TO POINTER
    CUDACHECK(hipMalloc(&sendBuffer_d,sizeof(transferPoints*)*nRanks));
    for(int i = 0; i < nRanks; i++)
    {
      transferPoints *darray;
      CUDACHECK(hipMalloc(&darray, sizeof(transferPoints) * sendPoints[i]));
      CUDACHECK(hipMemcpy(darray,sendBuffer[i], sendPoints[i] * sizeof(transferPoints), hipMemcpyHostToDevice));
      CUDACHECK(hipMemcpy(&sendBuffer_d[i], &darray, sizeof(transferPoints*), hipMemcpyHostToDevice));
      memcpy(&sendPointer[i],&darray,sizeof(transferPoints*));
    }

    CUDACHECK(hipMalloc(&receiveBuffer_d,sizeof(transferPoints*)*nRanks));
    for(int i = 0; i < nRanks; i++)
    {
      transferPoints *darray;
      CUDACHECK(hipMalloc(&darray, sizeof(transferPoints) * receivePoints[i]));
      CUDACHECK(hipMemcpy(&receiveBuffer_d[i], &darray, sizeof(transferPoints*), hipMemcpyHostToDevice));
      memcpy(&receivePointer[i],&darray,sizeof(transferPoints*));
    }

    CUDACHECK(hipMalloc(&globalToGhostIndex_send, nRanks * sizeof(int*)));
    for(int i = 0; i < nRanks; i++)
    {
      int *darray;
      CUDACHECK(hipMalloc(&darray, sizeof(int) * max_points));
      CUDACHECK(hipMemcpy(darray,globalToGhostIndex[i], sizeof(int) * max_points, hipMemcpyHostToDevice));
      CUDACHECK(hipMemcpy(&globalToGhostIndex_send[i], &darray, sizeof(int*), hipMemcpyHostToDevice));
      memcpy(&globalToGhostIndexSendPointer[i],&darray,sizeof(int*));
    }

    CUDACHECK(hipMalloc(&globalToGhostIndex_receive, nRanks * sizeof(int*)));
    for(int i = 0; i < nRanks; i++)
    {
        int *darray;
        CUDACHECK(hipMalloc(&darray, sizeof(int) * max_points));
        CUDACHECK(hipMemcpy(&globalToGhostIndex_receive[i], &darray, sizeof(int*), hipMemcpyHostToDevice));
        memcpy(&globalToGhostIndexReceivePointer[i],&darray,sizeof(int*));
    }

    CUDACHECK(hipMalloc(&splitPoint_d, numberOfPointsPerDevice * sizeof(splitPoints)));
    CUDACHECK(hipMemcpy(splitPoint_d, splitPoint, numberOfPointsPerDevice * sizeof(splitPoints), hipMemcpyHostToDevice));
    CUDACHECK(hipMalloc(&globalToLocalIndex_temp, max_points * sizeof(int)));
    CUDACHECK(hipMemcpy(globalToLocalIndex_temp, globalToLocalIndex, max_points * sizeof(int), hipMemcpyHostToDevice));
    CUDACHECK(hipMalloc(&partVector_d, max_points * sizeof(int)));
    CUDACHECK(hipMemcpy(partVector_d, &partVector, max_points * sizeof(int), hipMemcpyHostToDevice));
    
    // //
   
    
    if (myRank == 0) {
      cout<<"Getting NCCL Unique ID\n";
      NCCLCHECK(ncclGetUniqueId(&id));
    }
    MPICHECK(MPI_Barrier(MPI_COMM_WORLD));

    MPICHECK(MPI_Bcast((void *)&id, sizeof(id), MPI_BYTE, 0, MPI_COMM_WORLD));
    // //
    if(myRank==0){
      cout<<"ID Succesfully Broadcasted\n";
    }

    // Initialising NCCL Communicator and CUDA stream
    ncclComm_t comm;
    NCCLCHECK(ncclCommInitRank(&comm, nRanks, id, myRank));
    hipStream_t stream;

    if(myRank==0){
      cout<<"Beginning Solver\n";
    }

    auto start = high_resolution_clock::now();
    // 
    fpi_solver_multi_nccl(splitPoint_d,localRank,sendBuffer_d,receiveBuffer_d,nRanks,myRank,sendPoints,receivePoints,comm,stream,sendPointer,receivePointer,globalToLocalIndex_temp,globalToGhostIndex_receive,globalToGhostIndexSendPointer,globalToGhostIndexReceivePointer,partVector_d);
    // 
    auto stop = high_resolution_clock::now();
    if(myRank==0){
      cout<<"Copying memory back to Host\n";
    }
    CUDACHECK(hipMemcpy(splitPoint, splitPoint_d, numberOfPointsPerDevice * sizeof(splitPoints), hipMemcpyDeviceToHost));

    
    // TO COPY BACK THE SEND BUFFER TO HOST (POINTER TO POINTER METHOD)
    // transferPoints *darray;
    // for (int i = 0; i <nRanks; i++){
    //   hipMalloc(&darray, sizeof(transferPoints) * sendPoints[i]);
    //   hipMemcpy(&darray, &sendBuffer_d[i], sizeof(transferPoints*), hipMemcpyDeviceToHost);
    //   hipMemcpy(sendBuffer[i], darray, sizeof(transferPoints) * sendPoints[i], hipMemcpyDeviceToHost);
    //   hipFree(darray);
    // }

    // for (int i = 0; i <nRanks; i++){
    //   hipMalloc(&darray, sizeof(transferPoints) * receivePoints[i]);
    //   hipMemcpy(&darray, &receiveBuffer_d[i], sizeof(transferPoints*), hipMemcpyDeviceToHost);
    //   hipMemcpy(receiveBuffer[i], darray, sizeof(transferPoints) * receivePoints[i], hipMemcpyDeviceToHost);
    // }

    if(myRank==0){
      cout<<"Deallocating memory and Destroying Communicators\n";
    }
    // 
    CUDACHECK(hipFree(splitPoint_d));
    // 
    NCCLCHECK(ncclCommDestroy(comm));
    MPI_Finalize();
    //
    auto duration = duration_cast<microseconds>(stop - start);
    cout << "Done with process "<<myRank<< ". Time Taken by was:" << duration.count() / 1000000.0 << endl;
}
