#include "hip/hip_runtime.h"
// #include <iostream>
// #include <fstream>
// #include <random>
// #include "split_fluxes_mod.h"
// #include <iomanip>
// #include "octant_fluxes_mod.h"
// #include "wall_flux_dGxneg_mod.h"
// #include "wall_flux_dGyneg_mod.h"
// #include "wall_flux_dGxpos_mod.h"
// #include "wall_flux_dGypos_mod.h"
// #include "wall_flux_dGzneg_mod.h"
// #include "point_preprocessor_mod.h"
// #include "compute_conserved_vector_mod.h"
// #include "timestep_delt_mod.h"
// #include "generate_connectivity_mod.h"
// #include "implicit_aliasing_mod.h"
// #include "flux_residual_mod.h"
// #include "initial_conditions_mod.h"
// #include "interior_flux_dGxneg_mod.h"
// #include "interior_flux_dGxpos_mod.h"
// #include "interior_flux_dGyneg_mod.h"
// #include "interior_flux_dGypos_mod.h"
// #include <chrono>
// #include <hip/hip_runtime.h>
// #include "fpi_solver_mod.h"
// #include <nccl.h>
// #include <mpi.h>

// // #include "cuPrintf.cu"
// using namespace std;
// using namespace std::chrono;

// #define MPICHECK(cmd) do {                          \
//     int e = cmd;                                      \
//     if( e != MPI_SUCCESS ) {                          \
//       printf("Failed: MPI error %s:%d '%d'\n",        \
//           __FILE__,__LINE__, e);   \
//       exit(EXIT_FAILURE);                             \
//     }                                                 \
//   } while(0)

// #define CUDACHECK(cmd) do {                         \
//     hipError_t err = cmd;                            \
//     if (err != hipSuccess) {                         \
//       printf("Failed: Cuda error %s:%d '%s'\n",       \
//           __FILE__,__LINE__,hipGetErrorString(err)); \
//       exit(EXIT_FAILURE);                             \
//     }                                                 \
//   } while(0)
  
  
//   #define NCCLCHECK(cmd) do {                         \
//     ncclResult_t res = cmd;                           \
//     if (res != ncclSuccess) {                         \
//       printf("Failed, NCCL error %s:%d '%s'\n",       \
//           __FILE__,__LINE__,ncclGetErrorString(res)); \
//       exit(EXIT_FAILURE);                             \
//     }                                                 \
//   } while(0)

//   int main(int argc, char* argv[])
//   {
  
//       cout<<setprecision(13)<<scientific;
  
//       read_input_point_data();
//       initial_conditions();
//       generate_split_stencils();
//       //
//       fstream fin;
//       fin.open("/home/anil/new_3d_code/3d-mfcfd/inputFiles/partFile.dat",ios::in);
//       fin>>numDevices;
//       int counter;
//       for(int i=0;i<max_points;i++){
//           fin>>point.counter[i]>>partVector[i];
//           numberOfPointsPerDevice[partVector[i]]++;
//       }
//       fin.close();
//       //
//       for(int i=0;i<numDevices;i++){
//           splitPoint[i]=new splitPoints[numberOfPointsPerDevice[i]];
//       }
//       int *pointCounter=new int[numDevices];
//       for(int i=0;i<numDevices;i++){
//           pointCounter[i]=0;
//       }
//       for(int i=0;i<max_points;i++){
//           assign(splitPoint[partVector[i]][pointCounter[partVector[i]]],i);
//           pointCounter[partVector[i]]++;
//       }
//       //
//       comms=new ncclComm_t[numDevices];
//       devs=new int[numDevices];
//       for(int i=0;i<numDevices;i++){
//           devs[i]=i;
//       }
//       cout<<"HI1\n";
//       int nGPUs;
//       hipGetDeviceCount(&nGPUs);
//       cout<<nGPUs<<endl;
//       //
//       splitPoints** splitPoint_d = (splitPoints**)malloc(numDevices * sizeof(splitPoints*));
//       hipStream_t *s = (hipStream_t*)malloc(sizeof(hipStream_t)*numDevices);
//       cout<<"HI2\n";
//       cout<<numDevices<<endl;
//       //
//       for (int i = 0; i < numDevices; ++i) {
//           CUDACHECK(hipSetDevice(i));
//           CUDACHECK(hipMalloc(splitPoint_d + i, numberOfPointsPerDevice[i] * sizeof(splitPoints)));
//           CUDACHECK(hipStreamCreate(&s[i]));
//           CUDACHECK(hipMemcpyAsync(splitPoint_d[i], splitPoint[i], numberOfPointsPerDevice[i] * sizeof(splitPoints), hipMemcpyHostToDevice, s[i]));
//       }
//       //
//       cout<<splitPoint[0][0].x<<" "<<splitPoint[1][1].x<<endl;
//       cout<<"HI3\n";
//       // if (myRank == 0) ncclGetUniqueId(&id);
//       // MPICHECK(MPI_Bcast((void *)&id, sizeof(id), MPI_BYTE, 0, MPI_COMM_WORLD));
    
    
//       // initializing NCCL, group API is required around ncclCommInitRank as it is
//       // called across multiple GPUs in each thread/process
//       // NCCLCHECK(ncclGroupStart());
//       // for (int i=0; i<numDevices; i++) {
//       //    CUDACHECK(hipSetDevice(localRank*numDevices + i));
//       //    NCCLCHECK(ncclCommInitRank(comms+i, nRanks*numDevices, id, myRank*numDevices + i));
//       // }
//       // NCCLCHECK(ncclGroupEnd());
  
//       for (int i=0; i<numDevices; i++)
//         CUDACHECK(hipStreamSynchronize(s[i]));
//       cout<<"HI4\n";
//       // //
//       fpi_solver_multi_nccl(splitPoint_d,s);
//       // //
//       cout<<"HI5\n";
//       //
//       for (int i = 0; i < numDevices; ++i) {
//           CUDACHECK(hipSetDevice(i));
//           CUDACHECK(hipMemcpyAsync(splitPoint[i], splitPoint_d[i], numberOfPointsPerDevice[i] * sizeof(splitPoints), hipMemcpyDeviceToHost, s[i]));
//       }
//       cout<<"HI6\n";
//       cout<<splitPoint[0][10].x<<endl;
//       cout<<"HI7\n";
//       //
//       for (int i=0; i<numDevices; i++) {
//         CUDACHECK(hipFree(splitPoint_d[i]));
//       }
//       //
//       for (int i=0; i<numDevices; ++i){
//           ncclCommDestroy(comms[i]);
//       }
//       cout << "Done with process "<<endl;
//   }


// // int main(int argc, char* argv[])
// // {

// //     cout<<setprecision(13)<<scientific;

// //     read_input_point_data();
// //     initial_conditions();
// //     generate_split_stencils();
// //     //
// //     fstream fin;
// //     fin.open("/home/anil/new_3d_code/3d-mfcfd/inputFiles/partFile.dat",ios::in);
// //     fin>>numDevices;
// //     int counter;
// //     for(int i=0;i<max_points;i++){
// //         fin>>point.counter[i]>>partVector[i];
// //         numberOfPointsPerDevice[partVector[i]]++;
// //     }
// //     fin.close();
// //     //
// //     // cout<<numberOfPointsPerDevice[0]<<" "<<numberOfPointsPerDevice[1]<<" "<<numberOfPointsPerDevice[0]+numberOfPointsPerDevice[1]<<endl;
// //     //
// //     for(int i=0;i<numDevices;i++){
// //         splitPoint[i]=new splitPoints[numberOfPointsPerDevice[i]];
// //     }
// //     int *pointCounter=new int[numDevices];
// //     for(int i=0;i<numDevices;i++){
// //         pointCounter[i]=0;
// //     }
// //     for(int i=0;i<max_points;i++){
// //         assign(splitPoint[partVector[i]][pointCounter[partVector[i]]],i);
// //         pointCounter[partVector[i]]++;
// //     }
// //     //
// //     //
// //     // int myRank, nRanks, localRank = 0;
  
  
// //     //initializing MPI
// //     // MPICHECK(MPI_Init(&argc, &argv));
// //     // MPICHECK(MPI_Comm_rank(MPI_COMM_WORLD, &myRank));
// //     // MPICHECK(MPI_Comm_size(MPI_COMM_WORLD, &nRanks));
// //     //
// //     comms=new ncclComm_t[numDevices];
// //     devs=new int[numDevices];
// //     for(int i=0;i<numDevices;i++){
// //         devs[i]=i;
// //     }
// //     cout<<"HI1\n";
// //     //
// //     splitPoints** splitPoint_d = (splitPoints**)malloc(numDevices * sizeof(splitPoints*));
// //     hipStream_t *s = (hipStream_t*)malloc(sizeof(hipStream_t)*numDevices);
// //     cout<<"HI2\n";
// //     cout<<numDevices<<endl;
// //     //
// //     for (int i = 0; i < numDevices; ++i) {
// //         CUDACHECK(hipSetDevice(i));
// //         CUDACHECK(hipMalloc(splitPoint_d + i, numberOfPointsPerDevice[i] * sizeof(splitPoints)));
// //         CUDACHECK(hipStreamCreate(&s[i]));
// //         CUDACHECK(hipMemcpyAsync(splitPoint_d[i], splitPoint[i], numberOfPointsPerDevice[i] * sizeof(splitPoints), hipMemcpyHostToDevice, s[i]));
// //     }
// //     //
// //     cout<<splitPoint[0][0].x<<" "<<splitPoint[1][1].x<<endl;
// //     cout<<"HI3\n";
// //     // if (myRank == 0) ncclGetUniqueId(&id);
// //     // MPICHECK(MPI_Bcast((void *)&id, sizeof(id), MPI_BYTE, 0, MPI_COMM_WORLD));
  
  
// //     // initializing NCCL, group API is required around ncclCommInitRank as it is
// //     // called across multiple GPUs in each thread/process
// //     // NCCLCHECK(ncclGroupStart());
// //     // for (int i=0; i<numDevices; i++) {
// //     //    CUDACHECK(hipSetDevice(localRank*numDevices + i));
// //     //    NCCLCHECK(ncclCommInitRank(comms+i, nRanks*numDevices, id, myRank*numDevices + i));
// //     // }
// //     // NCCLCHECK(ncclGroupEnd());

// //     for (int i=0; i<numDevices; i++)
// //       CUDACHECK(hipStreamSynchronize(s[i]));
// //     cout<<"HI4\n";
// //     // //
// //     fpi_solver_multi_nccl(splitPoint_d,s);
// //     // //
// //     cout<<"HI5\n";
// //     //
// //     for (int i = 0; i < numDevices; ++i) {
// //         CUDACHECK(hipSetDevice(i));
// //         CUDACHECK(hipMemcpyAsync(splitPoint[i], splitPoint_d[i], numberOfPointsPerDevice[i] * sizeof(splitPoints), hipMemcpyDeviceToHost, s[i]));
// //     }
// //     cout<<"HI6\n";
// //     cout<<splitPoint[0][10].x<<endl;
// //     cout<<"HI7\n";
// //     //
// //     for (int i=0; i<numDevices; i++) {
// //       CUDACHECK(hipFree(splitPoint_d[i]));
// //     }
// //     //
// //     for (int i=0; i<numDevices; ++i){
// //         ncclCommDestroy(comms[i]);
// //     }
// //     cout << "Done with process "<<endl;
// // }

// #include <nccl.h>
// #include <cstdio>
// #include <cstdlib>
// #include <iostream>
// using namespace std;
 
// __global__ void kernel(int *a,int g) 
// {
//   int index = threadIdx.x;

//   a[index] *= (g+1);
//   printf("%d\t", a[index]);

// }/*kernel*/
 

// void print_vector(int *in, int n){

//  for(int i=0; i < n; i++)
//   printf("%d\t", in[i]);

//  printf("\n");

// }/*print_vector*/


// int main(int argc, char* argv[]) {

//   int data_size = 8 ;
//   int nGPUs = 0;
//   hipGetDeviceCount(&nGPUs);
  
//   int *DeviceList = (int *) malloc (nGPUs     * sizeof(int));
//   int *data       = (int*)  malloc (data_size * sizeof(int));
//   int **d_data    = (int**) malloc (nGPUs     * sizeof(int*));
  
//   for(int i = 0; i < nGPUs; i++)
//       DeviceList[i] = i;
  
//       ncclUniqueId id;
//       ncclGetUniqueId(&id);
//       // printf("ID: %d\n",id);

//   /*Initializing NCCL with Multiples Devices per Thread*/
//   ncclComm_t* comms = (ncclComm_t*)  malloc(sizeof(ncclComm_t)  * nGPUs);  
//   hipStream_t* s   = (hipStream_t*)malloc(sizeof(hipStream_t)* nGPUs);
//   ncclGroupStart();
//   for(int i=0;i<nGPUs;i++){
//     hipSetDevice(DeviceList[i]);
//     ncclCommInitRank(comms, nGPUs, id,i);
//   }
//   ncclGroupEnd();
  
//   ncclResult_t asyncError;
//   ncclCommGetAsyncError(comms[0], &asyncError);
//   cout<<"Error :"<<asyncError<<endl;

//    int RanksinNCCLComm;
//    ncclCommCount( comms[1], &RanksinNCCLComm);
//    cout<<"RanksinNCCLComm "<<RanksinNCCLComm<<endl;

//    int AssociatedCudaDeviceRank;
//    for(int i=0;i<nGPUs;i++){
//     ncclCommCuDevice( comms[i], &AssociatedCudaDeviceRank);
//     cout<<"AssociatedCudaDeviceRank "<<AssociatedCudaDeviceRank<<endl;
//    }

//    int RankofComm;
//    ncclCommUserRank( comms[1], &RankofComm);
//    cout<<"RankofComm "<<RankofComm<<endl;
//   /*Population the data vector*/
//   // for(int i = 0; i < data_size; i++)
//   //     data[i] = rand()%(10-2)*2;
 
//   // // print_vector(data, data_size);
      
//   // for(int g = 0; g < nGPUs; g++) {
//   //     hipSetDevice(DeviceList[g]);
//   //     hipStreamCreate(&s[g]);
//   //     hipMalloc(&d_data[g], data_size * sizeof(int));
     
//   //     if(g == 0)  /*Copy from Host to Device*/
//   //        hipMemcpy(d_data[g], data, data_size * sizeof(int), hipMemcpyHostToDevice);
//   // }
        
//   // ncclGroupStart();
 
//   // 		for(int g = 0; g < nGPUs; g++) {
//   // 	  	    hipSetDevice(DeviceList[g]);
//   //   	  	    ncclBcast(d_data[g], data_size, ncclInt, 0, comms[g], s[g]); /*Broadcasting it to all*/
//   // 		}

//   // ncclGroupEnd();       

//   // for (int g = 0; g < nGPUs; g++) {
//   //     hipSetDevice(DeviceList[g]);
//   //     ncclCommCuDevice( comms[g], &currRank);
//   //     printf("Current Rank : %d\n",currRank);
//   //     printf("\nThis is device %d\n", g);
//   //     kernel <<< 1 , data_size >>> (d_data[g],g);/*Call the CUDA Kernel: The code multiple the vector position per 2 on GPUs*/
//   //     hipDeviceSynchronize();    
//   //     cout<<endl;         
//   // }

//   // printf("\n");

//   // for (int g = 0; g < nGPUs; g++) { /*Synchronizing CUDA Streams*/
//   //     hipSetDevice(DeviceList[g]);
//   //     hipStreamSynchronize(s[g]);
//   // }
 
//   // for(int g = 0; g < nGPUs; g++) {  /*Destroy CUDA Streams*/
//   //     hipSetDevice(DeviceList[g]);
//   //     hipStreamDestroy(s[g]);
//   // }

//   for(int g = 0; g < nGPUs; g++)    /*Finalizing NCCL*/
//      ncclCommDestroy(comms[g]);
  
//   /*Freeing memory*/
//   free(s);
//   free(data); 
//   free(DeviceList);

//   hipFree(d_data);

//   return 0;

// }/*main*/

//
// Multiple Devices per Thread
// 


//
// execute command: mpirun -np 2 ./ex3gather.out 
//
/* output result:
myRank: 0 localRank: 0
myRank: 1 localRank: 1
myRank0 sendbuff[0]
 j: 0 hptr[i][j]: 0
 j: 1 hptr[i][j]: 1
 j: 2 hptr[i][j]: 2
myRank1 sendbuff[0]
 j: 0 hptr[i][j]: 0
 j: 1 hptr[i][j]: 1
 j: 2 hptr[i][j]: 2
Root is:0 ncclgather result is :
 j: 0 hptr[i][j]: 0
 j: 1 hptr[i][j]: 1
 j: 2 hptr[i][j]: 2
 j: 3 hptr[i][j]: 0
 j: 4 hptr[i][j]: 1
 j: 5 hptr[i][j]: 2
[MPI Rank 0] Success 
[MPI Rank 1] Success
*/
#include <stdio.h>
#include "hip/hip_runtime.h"
#include "nccl.h"
#include "mpi.h"
// #include "ncclEnhance.h"
#include <unistd.h>
#include <stdint.h>
#include <iostream>

#define MPICHECK(cmd) do {                          \
  int e = cmd;                                      \
  if( e != MPI_SUCCESS ) {                          \
    printf("Failed: MPI error %s:%d '%d'\n",        \
        __FILE__,__LINE__, e);   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)


#define CUDACHECK(cmd) do {                         \
  hipError_t e = cmd;                              \
  if( e != hipSuccess ) {                          \
    printf("Failed: Cuda error %s:%d '%s'\n",             \
        __FILE__,__LINE__,hipGetErrorString(e));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)


#define NCCLCHECK(cmd) do {                         \
  ncclResult_t r = cmd;                             \
  if (r!= ncclSuccess) {                            \
    printf("Failed, NCCL error %s:%d '%s'\n",             \
        __FILE__,__LINE__,ncclGetErrorString(r));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)


static uint64_t getHostHash(const char* string) {
  // Based on DJB2a, result = result * 33 ^ char
  uint64_t result = 5381;
  for (int c = 0; string[c] != '\0'; c++){
    result = ((result << 5) + result) ^ string[c];
  }
  return result;
}


static void getHostName(char* hostname, int maxlen) {
  gethostname(hostname, maxlen);
  for (int i=0; i< maxlen; i++) {
    if (hostname[i] == '.') {
        hostname[i] = '\0';
        return;
    }
  }
}

__global__ void  init(float *dptr,int myRank)
{
  int id = threadIdx.x;
  dptr[id] = id;
}


int main(int argc, char* argv[])
{
    //each process is using two GPUs
    int nDev = 1;
    int root = 0;
    int size = 3;

    int myRank, nRanks, localRank = 0;

    //initializing MPI
    MPICHECK(MPI_Init(&argc, &argv));
    MPICHECK(MPI_Comm_rank(MPI_COMM_WORLD, &myRank));
    MPICHECK(MPI_Comm_size(MPI_COMM_WORLD, &nRanks));

    if (myRank == 0)
    {
        std::cout << "================================================================"
                  << "\n    Executing " << argv[0] << " now!\n"
                  << "================================================================\n";
    }

    //calculating localRank which is used in selecting a GPU
    uint64_t hostHashs[nRanks];
    char hostname[1024];
    getHostName(hostname, 1024);
    hostHashs[myRank] = getHostHash(hostname);
    MPICHECK(MPI_Allgather(MPI_IN_PLACE, 0, MPI_DATATYPE_NULL, hostHashs, sizeof(uint64_t), MPI_BYTE, MPI_COMM_WORLD));
    for (int p = 0; p < nRanks; p++)
    {
      if (p == myRank)
        break;
      if (hostHashs[p] == hostHashs[myRank])
        localRank++;
    }
    std::cout << "myRank: " << myRank << " localRank: " << localRank << "\n";

    float **sendbuff = (float **)malloc(nDev * sizeof(float *));
    float **recvbuff = (float **)malloc(nDev * sizeof(float *));
    float **hptr = (float **)malloc(nDev * sizeof(float *));
    hipStream_t *s = (hipStream_t *)malloc(sizeof(hipStream_t) * nDev);

    //picking GPUs based on localRank
    for (int i = 0; i < nDev; ++i)
    {
      CUDACHECK(hipSetDevice(localRank * nDev + i)); // 给所有设备编号
      CUDACHECK(hipMalloc(sendbuff + i, size * sizeof(float)));
      CUDACHECK(hipMalloc(recvbuff + i, nDev * nRanks * size * sizeof(float)));
      CUDACHECK(hipMemset(sendbuff[i], 1, size * sizeof(float)));
      CUDACHECK(hipMemset(recvbuff[i], 0, size * sizeof(float)));
      CUDACHECK(hipStreamCreate(s + i));
      hptr[i] = (float *)malloc(nDev * nRanks * size * sizeof(float));
  }


  ncclUniqueId id;
  ncclComm_t comms[nDev];


  //generating NCCL unique ID at one process and broadcasting it to all
  if (myRank == 0) ncclGetUniqueId(&id);
  MPICHECK(MPI_Bcast((void *)&id, sizeof(id), MPI_BYTE, 0, MPI_COMM_WORLD));

  //initializing NCCL, group API is required around ncclCommInitRank as it is
  //called across multiple GPUs in each thread/process
  NCCLCHECK(ncclGroupStart());
  for (int i = 0; i < nDev; i++)
  {
    CUDACHECK(hipSetDevice(localRank * nDev + i));
    init<<<1, size>>>(sendbuff[i], myRank);
    NCCLCHECK(ncclCommInitRank(comms + i, nRanks * nDev, id, myRank * nDev + i));
    hipMemcpy(hptr[i],sendbuff[i],size*sizeof(float),hipMemcpyDeviceToHost);
    std::cout<<"myRank"<<myRank<<" sendbuff["<<i<<"]"<<"\n";
    for(int j=0;j<size;++j){
        std::cout<<" j: "<<j<<" hptr[i][j]: "<<hptr[i][j]<<"\n";
    }
  }
  NCCLCHECK(ncclGroupEnd());


  // gather Data
  for(int i=0;i<nDev;++i){
    NCCLGather(sendbuff[i], size, ncclFloat, recvbuff[i], size, ncclFloat, root, comms[i], s[i]);
  }

  for (int i = 0; i < nDev; ++i)
  {
    if(myRank * nDev + i==root)
    {
      hipMemcpy(hptr[i], recvbuff[i], nDev * nRanks * size * sizeof(float), hipMemcpyDeviceToHost);
      std::cout << "Root is:" << root << " ncclgather result is :\n";
      for (int j = 0; j < nRanks * nDev * size; ++j)
      {
        std::cout << " j: " << j << " hptr[i][j]: " << hptr[i][j] << "\n";
      }
    }
  }

  //synchronizing on CUDA stream to complete NCCL communication
  for (int i=0; i<nDev; i++)
      CUDACHECK(hipStreamSynchronize(s[i]));


  //freeing device memory
  for (int i=0; i<nDev; i++) {
     CUDACHECK(hipFree(sendbuff[i]));
     CUDACHECK(hipFree(recvbuff[i]));
     free(hptr[i]);
  }


  //finalizing NCCL
  for (int i=0; i<nDev; i++) {
     ncclCommDestroy(comms[i]);
  }


  //finalizing MPI
  MPICHECK(MPI_Finalize());


  printf("[MPI Rank %d] Success \n", myRank);
  return 0;
}