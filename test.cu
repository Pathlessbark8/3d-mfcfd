#include <iostream>
#include <fstream>
#include <random>
#include "split_fluxes_mod.h"
#include <iomanip>
#include "octant_fluxes_mod.h"
#include "wall_flux_dGxneg_mod.h"
#include "wall_flux_dGyneg_mod.h"
#include "wall_flux_dGxpos_mod.h"
#include "wall_flux_dGypos_mod.h"
#include "wall_flux_dGzneg_mod.h"
#include "point_preprocessor_mod.h"
#include "compute_conserved_vector_mod.h"
#include "timestep_delt_mod.h"
#include "generate_connectivity_mod.h"
#include "implicit_aliasing_mod.h"
#include "flux_residual_mod.h"
#include "initial_conditions_mod.h"
#include "interior_flux_dGxneg_mod.h"
#include "interior_flux_dGxpos_mod.h"
#include "interior_flux_dGyneg_mod.h"
#include "interior_flux_dGypos_mod.h"
#include <chrono>
#include <hip/hip_runtime.h>
#include "fpi_solver_mod.h"
using namespace std;
using namespace std::chrono;
int main()
{

    cout<<setprecision(13)<<scientific;
    
    cout<<"Reading Points\n";
    read_input_point_data();
    cout<<"Setting Initial Conditions\n";
    initial_conditions();
    cout<<"Generating Stencil\n";
    generate_split_stencils();

    // cout<<"Testing"<<endl;
    // for (int j = 0; j < point.zpos_nbhs[21]; j++)
	// //
	// {
    //     int k = point.zpos_conn[21][j];
    //     cout<<21<<" "<<j<<" "<<k<<endl;
    // }
    
    points *point_d;
    unsigned long long point_size = sizeof(point);
    hipStream_t stream;
    hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    //
    hipMalloc(&point_d, point_size);
    hipMemcpy(point_d, &point, point_size, hipMemcpyHostToDevice);
    // for (int j = 0; j < point.zpos_nbhs[21]; j++)
	// //
	// {
    //     int k = point.zpos_conn[21][j];
    //     cout<<k<<endl;
    // }
    hipDeviceSynchronize();
    auto start = high_resolution_clock::now();
    cout << "Starting CUDA excecution\n";
    //
    cout << setprecision(13);
    fpi_solver_cuda(point_d,stream);
    //
    hipDeviceSynchronize();
    auto stop = high_resolution_clock::now();
    auto duration = duration_cast<microseconds>(stop - start);
    cout << "Time Taken :" << duration.count() / 1000000.0 << endl;
    
    hipMemcpy(&point, point_d, point_size, hipMemcpyDeviceToHost);
   
    // fstream fout;
    // fout.open("output_prim.dat", ios::out);
    // for(int i=0;i<max_points;++i){
    //     fout<<point.prim[i][0]<<" "<<point.prim[i][1]<<" "<<point.prim[i][2]<<" "<<point.prim[i][3]<<" "<<point.prim[i][4]<<endl;
    // }
    // fout.close();
    hipFree(point_d);
    //
    cout << "Done\n";
}