// #include <iostream>
// #include <fstream>
// #include <random>
// #include "split_fluxes_mod.h"
// #include <iomanip>
// #include "octant_fluxes_mod.h"
// #include "wall_flux_dGxneg_mod.h"
// #include "wall_flux_dGyneg_mod.h"
// #include "wall_flux_dGxpos_mod.h"
// #include "wall_flux_dGypos_mod.h"
// #include "wall_flux_dGzneg_mod.h"
// #include "point_preprocessor_mod.h"
// #include "compute_conserved_vector_mod.h"
// #include "timestep_delt_mod.h"
// #include "generate_connectivity_mod.h"
// #include "implicit_aliasing_mod.h"
// #include "flux_residual_mod.h"
// #include "initial_conditions_mod.h"
// #include "interior_flux_dGxneg_mod.h"
// #include "interior_flux_dGxpos_mod.h"
// #include "interior_flux_dGyneg_mod.h"
// #include "interior_flux_dGypos_mod.h"
// #include <chrono>
// #include <cuda_runtime.h>
// #include "fpi_solver_mod.h"
// #include <nccl.h>
// using namespace std;
// using namespace std::chrono;

// #define CUDACHECK(cmd) do {                         \
//     cudaError_t err = cmd;                            \
//     if (err != cudaSuccess) {                         \
//       printf("Failed: Cuda error %s:%d '%s'\n",       \
//           __FILE__,__LINE__,cudaGetErrorString(err)); \
//       exit(EXIT_FAILURE);                             \
//     }                                                 \
//   } while(0)
  
  
//   #define NCCLCHECK(cmd) do {                         \
//     ncclResult_t res = cmd;                           \
//     if (res != ncclSuccess) {                         \
//       printf("Failed, NCCL error %s:%d '%s'\n",       \
//           __FILE__,__LINE__,ncclGetErrorString(res)); \
//       exit(EXIT_FAILURE);                             \
//     }                                                 \
//   } while(0)


// int main()
// {

//     cout<<setprecision(13)<<scientific;

//     read_input_point_data();
//     initial_conditions();
//     generate_split_stencils();
//     //
//     fstream fin;
//     fin.open("/home/anil/new_3d_code/3d-mfcfd/inputFiles/partFile.dat",ios::in);
//     fin>>numDevices;
//     int counter;
//     for(int i=0;i<max_points;i++){
//         fin>>point.counter[i]>>partVector[i];
//         numberOfPointsPerDevice[partVector[i]]++;
//     }
//     fin.close();
//     //
//     // cout<<numberOfPointsPerDevice[0]<<" "<<numberOfPointsPerDevice[1]<<" "<<numberOfPointsPerDevice[0]+numberOfPointsPerDevice[1]<<endl;
//     //
//     for(int i=0;i<numDevices;i++){
//         splitPoint[i]=new splitPoints[numberOfPointsPerDevice[i]];
//     }
//     // //
//     int *pointCounter=new int[numDevices];
//     for(int i=0;i<numDevices;i++){
//         pointCounter[i]=0;
//     }
//     for(int i=0;i<max_points;i++){
//         assign(splitPoint[partVector[i]][pointCounter[partVector[i]]],i);
//         pointCounter[partVector[i]]++;
//     }
//     //
//     // comms=new ncclComm_t[numDevices];
//     ncclComm_t comms[2];
//     int devs[2]={0,1};
//     // for(int i=0;i<numDevices;i++){
//     //     devs[i]=i;
//     // }
//     cout<<"HI1\n";
    
//     splitPoints** sendbuff = (splitPoints**)malloc(numDevices * sizeof(splitPoints*));
//     splitPoints** recvbuff = (splitPoints**)malloc(numDevices * sizeof(splitPoints*));
//     cudaStream_t* s = (cudaStream_t*)malloc(sizeof(cudaStream_t)*numDevices);
//     cout<<"HI2\n";
//     cout<<numDevices<<endl;
//     for (int i = 0; i < numDevices; ++i) {
//         CUDACHECK(cudaSetDevice(i));
//         cout<<i<<endl;
//         CUDACHECK(cudaMalloc(sendbuff + i, numberOfPointsPerDevice[i] * sizeof(splitPoints)));
//         CUDACHECK(cudaMalloc(recvbuff + i, numberOfPointsPerDevice[i] * sizeof(splitPoints)));
//         // CUDACHECK(cudaMemset(sendbuff[i], 1, numberOfPointsPerDevice[i] * sizeof(splitPoints)));
//         // CUDACHECK(cudaMemset(recvbuff[i], 0, numberOfPointsPerDevice[i] * sizeof(splitPoints)));
//         // CUDACHECK(cudaStreamCreate(s+i));
//     }
//     cout<<"HI3\n";
//     NCCLCHECK(ncclCommInitAll(comms, numDevices, devs));
//     cout<<"HI4\n";
//     // points *point_d;
//     // unsigned long long point_size = sizeof(point);
//     // cudaStream_t stream;
//     // cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking);
//     // //
//     // cudaMalloc(&point_d, point_size);
//     // cudaMemcpy(point_d, &point, point_size, cudaMemcpyHostToDevice);
//     // // cudaDeviceSynchronize();
//     // auto start = high_resolution_clock::now();
//     // cout << "Starting CUDA excecution\n";
//     // //
//     // cout << setprecision(13);
//     // fpi_solver_cuda(point_d,stream);
//     // //
//     // cudaDeviceSynchronize();
//     // auto stop = high_resolution_clock::now();
//     // auto duration = duration_cast<microseconds>(stop - start);
//     // cout << "Time Taken :" << duration.count() / 1000000.0 << endl;
//     // //
//     // cudaMemcpy(&point, point_d, point_size, cudaMemcpyDeviceToHost);
//     // fstream fout;
//     // fout.open("output_prim.dat", ios::out);
//     // for(int i=0;i<max_points;++i){
//     //     fout<<point.prim[0][i]<<" "<<point.prim[1][i]<<" "<<point.prim[2][i]<<" "<<point.prim[3][i]<<" "<<point.prim[4][i]<<endl;
//     // }
//     // fout.close();
//     // cudaFree(point_d);
//     //
//     for (int i=0; i<4; i++){
//         ncclCommDestroy(comms[i]);
//     }
//     cout << "Done\n";
// }

#include <stdio.h>
#include "hip/hip_runtime.h"
#include "nccl.h"

#define CUDACHECK(cmd) do {                         \
  hipError_t err = cmd;                            \
  if (err != hipSuccess) {                         \
    printf("Failed: Cuda error %s:%d '%s'\n",       \
        __FILE__,__LINE__,hipGetErrorString(err)); \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)


#define NCCLCHECK(cmd) do {                         \
  ncclResult_t res = cmd;                           \
  if (res != ncclSuccess) {                         \
    printf("Failed, NCCL error %s:%d '%s'\n",       \
        __FILE__,__LINE__,ncclGetErrorString(res)); \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)


int main(int argc, char* argv[])
{
  ncclComm_t comms[4];


  //managing 4 devices
  int nDev = 2;
  int size = 32*1024*1024;
  int devs[2] = { 0, 1 };


  //allocating and initializing device buffers
  float** sendbuff = (float**)malloc(nDev * sizeof(float*));
  float** recvbuff = (float**)malloc(nDev * sizeof(float*));
  hipStream_t* s = (hipStream_t*)malloc(sizeof(hipStream_t)*nDev);


  for (int i = 0; i < nDev; ++i) {
    CUDACHECK(hipSetDevice(i));
    CUDACHECK(hipMalloc(sendbuff + i, size * sizeof(float)));
    CUDACHECK(hipMalloc(recvbuff + i, size * sizeof(float)));
    // CUDACHECK(cudaMemset(sendbuff[i], 1, size * sizeof(float)));
    // CUDACHECK(cudaMemset(recvbuff[i], 0, size * sizeof(float)));
    CUDACHECK(hipStreamCreate(s+i));
  }


  //initializing NCCL
  NCCLCHECK(ncclCommInitAll(comms, nDev, devs));


   //calling NCCL communication API. Group API is required when using
   //multiple devices per thread
//   NCCLCHECK(ncclGroupStart());
//   for (int i = 0; i < nDev; ++i)
//     NCCLCHECK(ncclAllReduce((const void*)sendbuff[i], (void*)recvbuff[i], size, ncclFloat, ncclSum,
//         comms[i], s[i]));
//   NCCLCHECK(ncclGroupEnd());


//   //synchronizing on CUDA streams to wait for completion of NCCL operation
//   for (int i = 0; i < nDev; ++i) {
//     CUDACHECK(cudaSetDevice(i));
//     CUDACHECK(cudaStreamSynchronize(s[i]));
//   }


//   //free device buffers
//   for (int i = 0; i < nDev; ++i) {
//     CUDACHECK(cudaSetDevice(i));
//     CUDACHECK(cudaFree(sendbuff[i]));
//     CUDACHECK(cudaFree(recvbuff[i]));
//   }


  //finalizing NCCL
  for(int i = 0; i < nDev; ++i)
      ncclCommDestroy(comms[i]);


  printf("Success \n");
  return 0;
}
