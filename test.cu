#include <iostream>
#include <fstream>
#include <random>
#include "split_fluxes_mod.h"
#include <iomanip>
#include "octant_fluxes_mod.h"
#include "wall_flux_dGxneg_mod.h"
#include "wall_flux_dGyneg_mod.h"
#include "wall_flux_dGxpos_mod.h"
#include "wall_flux_dGypos_mod.h"
#include "wall_flux_dGzneg_mod.h"
#include "point_preprocessor_mod.h"
#include "compute_conserved_vector_mod.h"
#include "timestep_delt_mod.h"
#include "generate_connectivity_mod.h"
#include "implicit_aliasing_mod.h"
#include "flux_residual_mod.h"
#include "initial_conditions_mod.h"
#include "interior_flux_dGxneg_mod.h"
#include "interior_flux_dGxpos_mod.h"
#include "interior_flux_dGyneg_mod.h"
#include "interior_flux_dGypos_mod.h"
#include <chrono>
#include <hip/hip_runtime.h>
#include "fpi_solver_mod.h"
using namespace std;
using namespace std::chrono;
int main()
{

    cout<<setprecision(13)<<scientific;
    
    cout<<"Reading Points\n";
    read_input_point_data();
    cout<<"Setting Initial Conditions\n";
    initial_conditions();
    cout<<"Generating Stencil\n";
    generate_split_stencils();

    // cout<<"Testing"<<endl;
    // for (int j = 0; j < point.zpos_nbhs[21]; j++)
	// //
	// {
    //     int k = point.zpos_conn[21][j];
    //     cout<<21<<" "<<j<<" "<<k<<endl;
    // }
    
    points *point_d;
    unsigned long long point_size = sizeof(point);
    hipStream_t stream;
    hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    //
    hipMalloc(&point_d, point_size);
    hipMemcpy(point_d, &point, point_size, hipMemcpyHostToDevice);
    // for (int j = 0; j < point.zpos_nbhs[21]; j++)
	// //
	// {
    //     int k = point.zpos_conn[21][j];
    //     cout<<k<<endl;
    // }
    hipDeviceSynchronize();
    auto start = high_resolution_clock::now();
    cout << "Starting CUDA excecution\n";
    //
    cout << setprecision(13);
    fpi_solver_cuda(point_d,stream);
    //
    hipDeviceSynchronize();
    auto stop = high_resolution_clock::now();
    auto duration = duration_cast<microseconds>(stop - start);
    cout << "Time Taken :" << duration.count() / 1000000.0 << endl;
    
    hipMemcpy(&point, point_d, point_size, hipMemcpyDeviceToHost);
   
    if(restart){
        cout<<"Writing Restart File\n";
        fstream fout,fout2,fout3,fout4;
        fout.open("output_prim.dat", ios::out);
        fout2.open("output_q.dat", ios::out);
        fout3.open("output_qder.dat", ios::out);
        fout4.open("output_fluxres.dat", ios::out);
		fout<<setprecision(15);
		fout2<<setprecision(15);
        fout3<<setprecision(15);
        fout4<<setprecision(15);
        for(int i=0;i<max_points;++i){
            for(int r=0;r<5;++r){
                if(point.prim[i][r]<2e-13){
                    point.prim[i][r]=0;
                }
            }
        }
        for(int i=0;i<max_points;++i){
            fout<<point.prim[i][0]<<" "<<point.prim[i][1]<<" "<<point.prim[i][2]<<" "<<point.prim[i][3]<<" "<<point.prim[i][4]<<endl;
            fout2<<point.q[i][0]<<" "<<point.q[i][1]<<" "<<point.q[i][2]<<" "<<point.q[i][3]<<" "<<point.q[i][4]<<endl;
            fout3<<point.dq[i][0][0]<<" "<<point.dq[i][0][1]<<" "<<point.dq[i][0][2]<<" "<<point.dq[i][0][3]<<" "<<point.dq[i][0][4]<<endl;
            fout3<<point.dq[i][1][0]<<" "<<point.dq[i][1][1]<<" "<<point.dq[i][1][2]<<" "<<point.dq[i][1][3]<<" "<<point.dq[i][1][4]<<endl;
            fout3<<point.dq[i][2][0]<<" "<<point.dq[i][2][1]<<" "<<point.dq[i][2][2]<<" "<<point.dq[i][2][3]<<" "<<point.dq[i][2][4]<<endl;
            fout4<<point.flux_res[i][0]<<" "<<point.flux_res[i][1]<<" "<<point.flux_res[i][2]<<" "<<point.flux_res[i][3]<<" "<<point.flux_res[i][4]<<endl;
        }
        fout2.close();
        fout.close();
        fout3.close();
        fout4.close();
    }
    
    hipFree(point_d);
    //
    cout << "Done\n";
}